#include "hip/hip_runtime.h"
/*a*****************************************************
a*         2D Quasi Acoustic VTI Medium  FD & RTM
b*  P + sv wave and get rid of sv
c*  GPU(CUDA) ,poynting adcigs, read shot
d*
e*******************************************************
f*
g* Ps:  the Quasi Acoustic VTI function:
h*
i*          du/dt=1/rho*dp/dx ,
j*          dw/dt=1/rho*dq/dz ,
k*          dp/dt=rho*vpx^2*du/dx+rho*vp*vpn*dw/dz ,
l*          dq/dt=rho*vp*vpn*du/dx+rho*vp^2*dw/dz ,
m*                     vpx^2=vp^2*(1+2*epsilon);
n*                     vpn^2=vp^2*(1+2*delta);
o*
p*******************************************************
q*                           initial: 2017.02 Rong Tao
r*                            adcigs: 2017.04 Rong Tao
s*                            modify: 2018.02 Rong Tao
u*                             print: 2018.05 Rong Tao
v*
w*
x*
y*******************************************************
z*/

#include <stdio.h>
#include <malloc.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#define ANSI_CYAN    "\x1b[36m"
#define ANSI_GREEN   "\x1b[32m"
#define ANSI_RED     "\x1b[31m"
#define ANSI_YELLOW  "\x1b[33m"
#define ANSI_RESET   "\x1b[0m"

/*print error message*/
void print_error(const char *message)
{
    if(message)
        printf(ANSI_RED "     [Error]  " ANSI_RESET "%s",message);
}
/*print warning message*/
void print_warning(const char *message)
{
    if(message )
        printf(ANSI_YELLOW "  [Warning]  "ANSI_RESET "%s",message);
}
/*printf info message*/
void print_info(const char *message)
{
    if(message )
        printf(ANSI_CYAN "     [Info]  "ANSI_RESET "%s", message);
}
/*print success message*/
void print_success(const char* message)
{
    if(message )
        printf(ANSI_GREEN "  [Success]  "ANSI_RESET "%s", message);
}
#ifdef pi
#pragma message("Already define pi !!!")
#else
#define pi 3.141592653
#endif

#define mm 4

#define Nbar 25

#define CHECK_gpu(call)  {                                          \
    const hipError_t error = call;                             \
    char msg[1024];            \
    if (error != hipSuccess)  {                                \
        sprintf(msg, "Error in: %s:%d: ", __FILE__, __LINE__);  \
        print_error(msg);                                       \
        sprintf(msg, "code: %d, reason: %s\n", error,        \
                hipGetErrorString(error));                     \
        print_error(msg);                                       \
        exit(1);                                                \
    }                                            \
}

const char *note[] = {
"\n\033[1;31;44m       2D Quasi Acoustic VTI Medium  FD & RTM (CUDA, ADCIGs)     \033[m",
"\033[1;31;44m                               Author: Rong Tao @UPC             \033[m",
"                                                                               ",
" \033[1;31mQuasi Acoustic Function as follows:\033[m           ",
"    du/dt = dp/dx                                                       ",
"    dw/dt = dq/dz                                                       ",
"    dp/dt =  vpx^2 * du/dx + vp*vpn * dw/dz                             ",
"    dq/dt = vp*vpn * du/dx +  vp^2  * dw/dz                             ",
"    vpx^2 = vp^2 * (1+2*epsilon)                                        ",
"    vpn^2 = vp^2 * (1+2*delta)                                          ",
"                                                                               ",
" \033[1;31mRequired Parameters:\033[m                               ",
"    kind         =1 Finite difference forward modeling[FD]                       ",
"                 =2 Reverse Time Migration[RTM]                              ",
"    For example:                                                          ",
"    ./a.out  kind=1     Finite difference forward modeling[FD]                   ",
"    ./a.out  kind=2     Reverse Time Migration[RTM]                              ",
"                         no space in 'kind=1'                            ",
"                                                                               ",
" \033[1;31mInner Parameters:\033[m                                    ",
"    nx, dx       Horizontal Space sampling point and interval             ",
"    nz, dz       Vertical Space sampling point and interval               ",
"    nt, dt       Time sampling point and interval                         ",
"    favg         Wavelet frequency                                        ",
"    pfac         Wavelet Gain                                             ",
"    ns           The number of shots                                      ",
"    fs           First shot position[grid]                                ",
"    ds           Shots interval[grid]                                     ",
"    zs           Shots vertical position[grid]                            ",
"    nangle       The number of ADCIGs's angle                             ",
"    dangle       The interval of ADCIGs's angle                           ",
"    dAdcigs      Output file, the interval cdp(nx)                        ",
"    npml         PML Border width[grid]                                   ",
"                                                                               ",
" \033[1;31mOptional Parameters:\033[m                                   ",
"    wtype        kind of wavelet    =1 ricker wavelet                     ",
"                                    =2 derivative of gaussian             ",
"                                    =3 derivative of gaussian             ",
"    readShot     =true,             boolean, read obs shot                ",
"                 =false,            boolean, use accurate shot data       ",
"    writeSnap    =true,false        output snap into file or not          ",
"    ",
" \033[1;31mCOMMENTS\033[m                                                ",
"    Copyright (C) 2016-2018                                               ",
"            \033[1;32mChina University of Petroleum (East China), QingDao, China\033[m  ",
"    Authors: \033[1;32mRong Tao \033[m        ",
"    Location: \033[1;32mYi Fu Building @UPC\033[m                                        ",
"    Team: \033[1;32mLEON-VEL\033[m                                                       ",
"         ",
NULL
};


__device__ float d0;

__global__ void get_d0(float dx,
                       float dz,
                       int nnx,
                       int nnz,
                       int npml,
                       float *vp)
/* this (d0) function for pml bndr */
{
    d0 = 10.0*vp[nnx*nnz/2]*log(100000.0)/(2.0*npml*((dx+dz)/2.0));
}
/*#define mm 4*/
__constant__ float c[mm]={1.196289,-0.0797526,0.009570313,-0.0006975447};

void mBar(float fBar)
/* show progress bar */
{

    int i,j,k,m;
    //for ( i=0;i< Nbar+6; i++ )
    //    printf("\b");
    k = Nbar*fBar;
    m = fBar*100;
    printf("[");
    for ( i=0;i<k;i++ )
        printf("=");
    for ( j=0;j<Nbar-k;j++ )
        printf(" ");
    printf("]%3d%%",m);
}

void check_gpu_error (const char *msg)
/* check GPU errors */
{
    hipError_t err = hipGetLastError ();
    char message[1024];
    if (hipSuccess != err) {
        sprintf(message,"Cuda error: %s: %s\n", msg, hipGetErrorString(err));
        print_error(message);
        exit(0);
    }
    print_success("Check gpu successfully.\n");
}

void laplace_filter(int adj,
                    int nz,
                    int nx,
                    float *in,
                    float *out)
/**
 * linear operator
 *
 * Copyright@ Madagascar Mlaplac2
 */
{
    int iz,ix,j;
    for (j=0;j<nx*nz;j++)
        out[j]=0.0;

    for (ix=0; ix < nx; ix++) {
        for (iz=0; iz < nz; iz++) {
            j = iz+ix*nz;
            if (iz > 0) {
                if (adj) {
                    out[j-1] -= in[j];
                    out[j]   += in[j];
                } else {
                    out[j] += in[j] - in[j-1];
                }
            }
            if (iz < nz-1) {
                if (adj) {
                    out[j+1] -= in[j];
                    out[j]   += in[j];
                } else {
                    out[j] += in[j] - in[j+1];
                }
            }
            if (ix > 0) {
                if (adj) {
                    out[j-nz] -= in[j];
                    out[j]    += in[j];
                } else {
                    out[j] += in[j] - in[j-nz];
                }
            }
            if (ix < nx-1) {
                if (adj) {
                    out[j+nz] -= in[j];
                    out[j]    += in[j];
                } else {
                    out[j] += in[j] - in[j+nz];
                }
            }
        }
    }
}

__global__ void add_source( float pfac,
                            float xsn,
                            float zsn,
                            int nx,
                            int nz,
                            int nnx,
                            int nnz,
                            float dt,
                            float t,
                            float favg,
                            int wtype,
                            int npml,
                            int is,
                            int ds,
                            float *P,
                            float *Q)
/* generate ricker wavelet with time deley */
{
    int ixs,izs;
    float x_,xx_,tdelay,ts,source=0.0,fs;

    tdelay = 1.0/favg;
    ts = t-tdelay;
    fs = xsn+(is-1)*ds;

    if(wtype==1)//ricker wavelet
    {
        x_ = favg*ts;
        xx_ = x_*x_;
        source=(1-2*pi*pi*(xx_))*exp(-(pi*pi*xx_));

    }else if(wtype==2){//derivative of gaussian

        x_ = (-4)*favg*favg*pi*pi/log(0.1);
        source = (-2)*pi*pi*ts*exp(-x_*ts*ts);

    }else if(wtype==3){//derivative of gaussian

        x_ = (-1)*favg*favg*pi*pi/log(0.1);
        source = exp(-x_*ts*ts);
    }

    if(t <= 2*tdelay)
    {
        ixs = (int)( fs + 0.5) + npml - 1;
        izs = (int)(zsn + 0.5) + npml - 1;

        P[ixs*nnz+izs] += pfac * source;
        Q[ixs*nnz+izs] += pfac * source;
    }
}

__global__ void update_vel(int nx,
                           int nz,
                           int nnx,
                           int nnz,
                           int npml,
                           float dt,
                           float dx,
                           float dz,
                           float *u0,
                           float *w0,
                           float *u1,
                           float *w1,
                           float *P,
                           float *Q,
                           float *coffx1,
                           float *coffx2,
                           float *coffz1,
                           float *coffz2)
{
    int id=threadIdx.x+blockDim.x*blockIdx.x;

    int ix,iz,im;
    float dtx,dtz,xx,zz;

    ix = id/nnz;
    iz = id%nnz;

    dtx = dt/dx;
    dtz = dt/dz;

    if(id >= mm && id < nnx*nnz - mm) {

        if(ix >= mm && ix<(nnx-mm) && iz >= mm && iz<(nnz-mm)) {

            xx = 0.0;
            zz = 0.0;
            for(im = 0;im<mm;im++) {

                xx += c[im] * (P[id+(im+1)*nnz]  -  P[id-im*nnz]);
                zz += c[im] * (Q[id+im+1]        -  Q[id-im]);
            }
            u1[id] = coffx2[ix]*u0[id] - coffx1[ix]*dtx*xx;
            w1[id] = coffz2[iz]*w0[id] - coffz1[iz]*dtz*zz;
        }
    }
}

__global__ void update_stress(int nx,
                              int nz,
                              int nnx,
                              int nnz,
                              float dt,
                              float dx,
                              float dz,
                              float *u1,
                              float *w1,
                              float *P,
                              float *Q,
                              float *vp,
                              int npml,
                              float *px1,
                              float *px0,
                              float *pz1,
                              float *pz0,
                              float *qx1,
                              float *qx0,
                              float *qz1,
                              float *qz0,
                              float *acoffx1,
                              float *acoffx2,
                              float *acoffz1,
                              float *acoffz2,
                              float *delta,
                              float *epsilon,
                              int fs,
                              int ds,
                              int zs,
                              int is,
                              bool SV)
{
    int id=threadIdx.x+blockDim.x*blockIdx.x;

    int im, ix, iz, rx, rz;
    float dtx, dtz, xx, zz, ee, dd;

    /* iso circle */
    int R=18,r=7;

    ix = id / nnz;
    iz = id % nnz;

    dtx = dt / dx;
    dtz = dt / dz;

    if(id >= mm && id<nnx*nnz-mm) {

        /* iso circle begin */
        rx = ix-(fs+(is-1)*ds+npml);
        rz = iz-(zs+npml);

        if(SV){

            if((rx*rx+rz*rz) <= R*R){
                if((rx*rx+rz*rz) <= r*r){

                    ee = 0.0;
                    dd = 0.0;

                }else{

                    ee = 0.5*(1-cos(pi*((sqrtf(rx*rx+rz*rz)-r)*4.0/(R*3.0-1))))*epsilon[id];
                    dd = 0.5*(1-cos(pi*((sqrtf(rx*rx+rz*rz)-r)*4.0/(R*3.0-1))))*delta[id];

                }//else

            }else{

                    ee = epsilon[id];
                    dd = delta[id];
            }

        }else{

            ee = epsilon[id];
            dd = delta[id];

        }
        /* iso circle end */

        if(ix>=mm && ix<(nnx-mm) && iz>=mm && iz<(nnz-mm)) {

            xx=0.0;
            zz=0.0;

            for(im=0; im<mm; im++) {

                xx += c[im]*(u1[id+im*nnz] - u1[id-(im+1)*nnz]);
                zz += c[im]*(w1[id+im]     - w1[id-im-1]);
            }
            px1[id] = acoffx2[ix]*px0[id] - acoffx1[ix]*vp[id]*vp[id]*(1+2*ee)*dtx*xx;
            pz1[id] = acoffz2[iz]*pz0[id] - acoffz1[iz]*vp[id]*vp[id]*sqrtf(1+2*dd)*dtz*zz;
            qx1[id] = acoffx2[ix]*qx0[id] - acoffx1[ix]*vp[id]*vp[id]*sqrtf(1+2*dd)*dtx*xx;
            qz1[id] = acoffz2[iz]*qz0[id] - acoffz1[iz]*vp[id]*vp[id]*dtz*zz;

            P[id] = px1[id] + pz1[id];
            Q[id] = qx1[id] + qz1[id];
        }
    }
}

/**
 * Expand the border
 */
void pad_vv(int nx,
            int nz,
            int nnx,
            int nnz,
            int npml,
            float *ee)
{
    int ix,iz,id;

    for(id=0; id<nnx*nnz; id++) {

        ix = id/nnz;
        iz = id%nnz;

        /* left */
        if(ix<npml){

            ee[id] = ee[npml*nnz+iz];

        /* right */
        }else if(ix>=nnx-npml){

            ee[id] = ee[(nnx-npml-1)*nnz+iz];
        }
    }
    for(id=0; id<nnx*nnz; id++) {

        ix = id/nnz;
        iz = id%nnz;

        /* up */
        if(iz < npml){

            ee[id] = ee[ix*nnz+npml];

        /* bottom */
        }else if(iz >= nnz-npml){

            ee[id] = ee[ix*nnz+nnz-npml-1];
        }
    }
}

/**
 * Calculate the PML coefficient
 */
__global__ void initial_coffe(float dt,
                              int nn,
                              float *coff1,
                              float *coff2,
                              float *acoff1,
                              float *acoff2,
                              int npml)
{
    int id=threadIdx.x+blockDim.x*blockIdx.x;

    if(id < nn+2*npml) {

        /* The front of the inner */
        if(id<npml) {

            coff1[id] = 1.0/(1.0+(dt*d0*pow((npml-0.5-id)/npml,2.0))/2.0);
            coff2[id] = coff1[id]*(1.0-(dt*d0*pow((npml-0.5-id)/npml,2.0))/2.0);

            acoff1[id] = 1.0/(1.0+(dt*d0*pow(((npml-id)*1.0)/npml,2.0))/2.0);
            acoff2[id] = acoff1[id]*(1.0-(dt*d0*pow(((npml-id)*1.0)/npml,2.0))/2.0);

        /* media inner */
        }else if(id>=npml&&id<npml+nn){

            coff1[id] = 1.0;
            coff2[id] = 1.0;

            acoff1[id] = 1.0;
            acoff2[id] = 1.0;

        /* The tail of the inner */
        }else{

            coff1[id] = 1.0/(1.0+(dt*d0*pow((0.5+id-nn-npml)/npml,2.0))/2.0);
            coff2[id] = coff1[id]*(1.0-(dt*d0*pow((0.5+id-nn-npml)/npml,2.0))/2.0);

            acoff1[id] = 1.0/(1.0+(dt*d0*pow(((id-nn-npml)*1.0)/npml,2.0))/2.0);
            acoff2[id] = acoff1[id]*(1.0-(dt*d0*pow(((id-nn-npml)*1.0)/npml,2.0))/2.0);
        }
    }
}

/**
 * Record or load Receiver wavefield
 *      (nx) >> (nx,nt)
 *           or
 *   (nx,nt) >> (nx)
 */
__global__ void shot_record(int nnx,
                            int nnz,
                            int nx,
                            int nz,
                            int npml,
                            int it,
                            int nt,
                            float *P,
                            float *shot,
                            bool record)
{
    int id=threadIdx.x+blockDim.x*blockIdx.x;

    if(id<nx) {

        /* record the wavefield */
        if(record){

            shot[it+nt*id] = P[npml+nnz*(id+npml)];

        /* load the receiver wavefield */
        }else{

            P[npml+nnz*(id+npml)] = shot[it+nt*id];
        }
    }
}

/**
 * Record or backword the boundary wave field
 *
 */
__global__ void wavefield_bndr(int nnx,
                               int nnz,
                               int nx,
                               int nz,
                               int npml,
                               int it,
                               int nt,
                               float *P,
                               float *Q,
                               float *P_bndr,
                               float *Q_bndr,
                               bool record)
{
    int id=threadIdx.x+blockDim.x*blockIdx.x;

    if(id<2*nx+2*nz) {

        /* save boundary */
        if(record) {

            /* up */
            if(id<nx){

                P_bndr[it*(2*nx+2*nz)+id] = P[npml-1+nnz*(id+npml)];
                Q_bndr[it*(2*nx+2*nz)+id] = Q[npml-1+nnz*(id+npml)];

            /* bottom */
            }else if(id>=nx&&id<(2*nx)){

                P_bndr[it*(2*nx+2*nz)+id] = P[npml+nz+1+nnz*(id-nx+npml)];
                Q_bndr[it*(2*nx+2*nz)+id] = Q[npml+nz+1+nnz*(id-nx+npml)];

            /* left */
            }else if(id>=(2*nx)&&id<(2*nx+nz)){

                P_bndr[it*(2*nx+2*nz)+id] = P[id-2*nx+npml+nnz*(npml-1)];
                Q_bndr[it*(2*nx+2*nz)+id] = Q[id-2*nx+npml+nnz*(npml-1)];

            /* right */
            }else if(id>=(2*nx+nz)){

                P_bndr[it*(2*nx+2*nz)+id] = P[id-2*nx-nz+npml+nnz*(npml+nx+1)];
                Q_bndr[it*(2*nx+2*nz)+id] = Q[id-2*nx-nz+npml+nnz*(npml+nx+1)];

            }

        /* backward porpagation boundary */
        }else{

            /* up */
            if(id<nx){

                P[npml-1+nnz*(id+npml)] = P_bndr[it*(2*nx+2*nz)+id];
                Q[npml-1+nnz*(id+npml)] = Q_bndr[it*(2*nx+2*nz)+id];

            /* bottom */
            }else if(id>=nx&&id<(2*nx)){

                P[npml+nz+1+nnz*(id-nx+npml)] = P_bndr[it*(2*nx+2*nz)+id];
                Q[npml+nz+1+nnz*(id-nx+npml)] = Q_bndr[it*(2*nx+2*nz)+id];

            /* left */
            }else if(id>=(2*nx)&&id<(2*nx+nz)){

                P[id-2*nx+npml+nnz*(npml-1)] = P_bndr[it*(2*nx+2*nz)+id];
                Q[id-2*nx+npml+nnz*(npml-1)] = Q_bndr[it*(2*nx+2*nz)+id];

            /* right */
            }else if(id>=(2*nx+nz)){

                P[id-2*nx-nz+npml+nnz*(npml+nx+1)] = P_bndr[it*(2*nx+2*nz)+id];
                Q[id-2*nx-nz+npml+nnz*(npml+nx+1)] = Q_bndr[it*(2*nx+2*nz)+id];

            }
        }
    }
}

/**
 *mute direct waves
 */
__global__ void mute_directwave(int nx,
                                int nt,
                                float dt,
                                float favg,
                                float dx,
                                float dz,
                                int fs,
                                int ds,
                                int zs,
                                int is,
                                float *vp,
                                float *epsilon,
                                float *shot,
                                int tt)
{
    int it = threadIdx.x+blockDim.x*blockIdx.x;

    int mu_t, mu_nt;
    float mu_x, mu_z, mu_t0;

    int ix, id;

    for(ix = 0; ix < nx; ix ++){

        id = ix*nt + it;

        mu_x = dx*abs(ix-fs-(is-1)*ds);
        mu_z = dz*zs;
        mu_t0 = sqrtf(pow(mu_x,2)+pow(mu_z,2))/(vp[1]*sqrtf(1+2*epsilon[1]));
        mu_t = (int)(2.0/(dt*favg));
        mu_nt = (int)(mu_t0/dt)+mu_t+tt;

        if((it > (int)(mu_t0/dt)-tt) && (it<mu_nt))
            shot[id] = 0.0;
    }
}

/**
 * illumination matrix
 */
__global__ void cal_illumination(int nnx,
                                 int nnz,
                                 int nz,
                                 int npml,
                                 float *illumination,
                                 float *P,
                                 float *Q)
{
    int id = threadIdx.x+blockDim.x*blockIdx.x;
    int ix = id/nz;
    int iz = id%nz;

    if(id < nnx*nnz) {

        illumination[id] += P[iz+npml+nnz*(ix+npml)] * P[iz+npml+nnz*(ix+npml)]
                           +Q[iz+npml+nnz*(ix+npml)] * Q[iz+npml+nnz*(ix+npml)];

        if(illumination[id] <= 0.0 )
            illumination[id] = 1.0;
    }
}

/**
 * RTM migration
 */
__global__ void cal_migration(int nnx,
                              int nnz,
                              int nz,
                              int npml,
                              float *migration,
                              float *s,
                              float *g)
{
    int id = threadIdx.x+blockDim.x*blockIdx.x;
    int ix = id/nz;
    int iz = id%nz;

    if(id<nnx*nnz) {

        migration[id] += s[iz+npml+nnz*(ix+npml)] * g[iz+npml+nnz*(ix+npml)];
    }
}

/**
 *  illuminate
 */
__global__ void migration_illum(int nx,
                                int nz,
                                int npml,
                                float *migration,
                                float *illumination)
{
    int id=threadIdx.x+blockDim.x*blockIdx.x;

    if(id<nx*nz) {

        migration[id] /= illumination[id];
    }
}

/**
 *  poynting vector extraction ADCIGs
 *    "Copyright(C) Madagascar:user/pyang/Mrtmadcig.c"
 */
__global__ void Poynting_Adcigs(int nnz,
                                int nx,
                                int nz,
                                int npml,
                                int nangle,
                                int dangle,
                                float *adcigs,
                                float *s_P,
                                float *s_Q,
                                float *s_u,
                                float *s_w,
                                float *g_P,
                                float *g_Q,
                                float *g_u,
                                float *g_w)
{
    int id = threadIdx.x+blockDim.x*blockIdx.x;
    int ix = id/nz;
    int iz = id%nz;

    int ia = 0;

    float Ssx = -s_P[iz+npml+nnz*(ix+npml)]*s_u[iz+npml+nnz*(ix+npml)];
    float Ssz = -s_Q[iz+npml+nnz*(ix+npml)]*s_w[iz+npml+nnz*(ix+npml)];
    float Sgx =  g_P[iz+npml+nnz*(ix+npml)]*g_u[iz+npml+nnz*(ix+npml)];
    float Sgz =  g_Q[iz+npml+nnz*(ix+npml)]*g_w[iz+npml+nnz*(ix+npml)];

    float b1 =  Ssx*Ssx + Ssz*Ssz;
    float b2 =  Sgx*Sgx + Sgz*Sgz;
    float  a = (Ssx*Sgx + Ssz*Sgz)/(sqrtf(b1*b2)*(1 - 0.1));

    if(id<nx*nz) {

        if(a>=-1&&a<=1) {

          a = 0.5*acosf(a)*180.0/pi;
         ia = (int)(a/(dangle*1.0));

            if(ia<nangle) {
                adcigs[iz+nz*ia+nz*nangle*(id/nz)]
                    += s_P[iz+npml+nnz*(ix+npml)]*g_P[iz+npml+nnz*(ix+npml)]
                      *cosf(ia*pi/180.0)*cosf(ia*pi/180.0)*cosf(ia*pi/180.0);
            }
        }
    }
}

/**
 *  illuminate the adcigs
 */
__global__ void adcigs_illum(int nx,
                             int nz,
                             int nangle,
                             int dangle,
                             float *adcigs,
                             float *illumination)
{
    int id = threadIdx.x+blockDim.x*blockIdx.x;
    int ix = id/(nz*nangle);
    int iz = id%nz;

    if(id<nx*nz*nangle) {

        adcigs[id] /= illumination[iz+nz*ix];
    }
}

/**
 * Stack adcigs to migration
 * Can suppress low-frequency random noise
 */
void stk_adcigs(int nx,
                int nz,
                int nangle,
                float *adcigs,
                float *migration)
{
    int ix,iz,ia,id,ido;
    float stk;
    float *temp;

    temp=(float*)malloc(nz*nx*sizeof(float));

    for (ix=0; ix<nx; ix++)  {
        for (iz=0; iz<nz; iz++)  {
            stk=0.0;
            for (ia=0; ia<nangle; ia++)  {
                id = ix*nangle*nz+ia*nz+iz;
                stk += adcigs[id];
            }
            ido = ix*nz+iz;
            temp[ido] = stk;
        }
    }
    laplace_filter(1,nz,nx,temp,migration);
    print_success("Done with stack ADCIGs into Migration.\n");
}

/**
 * Draw thin adcigs
 */
void adcigs_smiled(int nx,
                   int nz,
                   int nangle,
                   int dAdcigs,
                   float *adcigs)
{
    int ix,iz,ia,id,ido;
    float *temp;

    temp = (float*)malloc(nz*nx/dAdcigs*nangle*sizeof(float));

    for (ix=0; ix<nx; ix++)  {
        for (ia=0; ia<nangle; ia++)  {
            for (iz=0; iz<nz; iz++)  {

                id=ix*nangle*nz+ia*nz+iz;

                if(ix%dAdcigs==0) {

                    ido = ix/dAdcigs*nangle*nz+ia*nz+iz;
                    temp[ido] = adcigs[id];
                    adcigs[ido] = temp[ido];
                }
            }
        }
    }
    print_success("Done with smile ADCIGs.\n");
}

void readFile( char FNvelocity[],
               char FNepsilon[],
               char FNdelta[],
               int nx,
               int nz,
               int nnx,
               int nnz,
               float dx,
               float dz,
               float favg,
               float dt,
               float *v,
               float *e,
               float *d,
               int npml)
{
    int i,j,id;
    float vmax, vmin;
    float emax, emin;
    float dmax, dmin;
    float H_min, dt_max, dxz_max, C, tmp;
    char msg[1024];
    FILE *fp1,*fp2,*fp3;


    if((fp1=fopen(FNvelocity,"rb"))==NULL){

        sprintf(msg, "error open <%s>!\n",FNvelocity);
        print_error(msg);
        exit(0);
    }
    if((fp2=fopen(FNepsilon,"rb"))==NULL){

        sprintf(msg, "error open <%s>!\n",FNepsilon);
        print_error(msg);
        exit(0);
    }
    if((fp3=fopen(FNdelta,"rb"))==NULL){

        sprintf(msg, "error open <%s>!\n",FNdelta);
        print_error(msg);
        exit(0);
    }

    vmin = emin = dmin =  999999.9;
    vmax = emax = dmax = -999999.9;

    for(i=npml;i<nx+npml;i++) {
        for(j=npml;j<nz+npml;j++) {

            id=i*nnz+j;
                                  /* inch time 0.3 */
            fread(&v[id],4L,1,fp1);//v[id] *= 0.3;
            fread(&e[id],4L,1,fp2);
            fread(&d[id],4L,1,fp3);

            /* For Parameters Sensitivity Analysis */
            //if(true) // true: active
           // if(v[id]>3800)
            {

                //v[id] *= 0.3;
                //e[id] *= 0.85;
                //d[id] *= 0.85;
            }

            if(vmax<v[id]) vmax = v[id];
            if(vmin>v[id]) vmin = v[id];
            if(emax<e[id]) emax = e[id];
            if(emin>e[id]) emin = e[id];
            if(dmax<d[id]) dmax = d[id];
            if(dmin>d[id]) dmin = d[id];
        }
    }
    fclose(fp1);
    fclose(fp2);
    fclose(fp3);

    print_success("Load (v,e,d) successfully.\n");

    sprintf(msg,"   Velocity Range (%.1f - %.1f)[m/s]\n",vmin,vmax);
    print_info(msg);
    sprintf(msg,"    Epsilon Range (%.4f - %.4f)\n",emin,emax);
    print_info(msg);
    sprintf(msg,"      Delta Range (%.4f - %.4f)\n",dmin,dmax);
    print_info(msg);

    /* boundary */
    pad_vv(nx,nz,nnx,nnz,npml,e);
    pad_vv(nx,nz,nnx,nnz,npml,d);
    pad_vv(nx,nz,nnx,nnz,npml,v);

    H_min=dx<dz?dx:dz;
    dt_max = 0.5*H_min/vmin;
    dxz_max = vmax/favg*0.2;

    if ( dxz_max<dz || dxz_max<dx){
        print_warning("You need have to redefine DX and DZ ! \n");
        exit(0);
    }
    if ( dt_max<dt){
        print_warning("You need have to redefine DT ! \n");
        exit(0);
    }
    if ( favg >= vmin/( 5.0*(dx>dz?dx:dz) )
      || favg >= vmin/( 5.0*(dx>dz?dx:dz) ) ) {
        print_warning("Non-dispersion relation not satisfied! \n");
        exit(0);
    }
    /* following
     * Copyright@ Madagascar */
    else if ( mm == 2 )
        C = 0.857;
    else if ( mm == 3 )
        C = 0.8;
    else if ( mm == 4 )
        C = 0.777;
    else if ( mm == 5 )
        C = 0.759;

    tmp = dt*vmax*sqrtf( 1.0/(dx*dx)+1.0/(dz*dz) );
    if ( tmp >= C){

        sprintf(msg, "Stability condition not satisfied! tmp = %f, C = %f\n",tmp,C);
        print_warning(msg);
        exit(0);
    }
}

/**
 * FD
 *         Author: Rong Tao
 */
void FD( char FNvelocity[],
         char FNepsilon[],
         char FNdelta[],
         char FNCalShot[],
         char FNSnap[],
         char FNIllumination[],
         int wtype,
         int npml,
         int nx,
         int nz,
         float dx,
         float dz,
         int nt,
         float dt,
         int ns,
         int fs,
         int ds,
         int zs,
         float favg,
         float pfac,
         bool writeSnap)
{
    float *v, *e, *d;
    float *vp, *epsilon, *delta;

    float *s_u0, *s_u1, *s_px0, *s_qx0, *s_px1, *s_qx1;
    float *s_w0, *s_w1, *s_pz0, *s_qz0, *s_pz1, *s_qz1;

    float *s_P, *s_Q;

    float *coffx1,*coffx2,*coffz1,*coffz2;
    float *acoffx1,*acoffx2,*acoffz1,*acoffz2;

    float *shot_Dev, *shot_Hos;
    float *illumination;

    int nnx, nnz;
    int it, is;

    float t;

    /* FILE pointer */
    FILE *fpCalShot = fopen(FNCalShot,"wb");
    FILE *fpSnap;
    if(writeSnap) {

        fpSnap = fopen(FNSnap,"wb");
    }
    FILE *fpIllunmination = fopen(FNIllumination,"wb");

    /* whole media size */
    nnx = nx + 2*npml;
    nnz = nz + 2*npml;

    /* read the media file into memory */
    v = (float*)malloc(nnz*nnx*sizeof(float));
    e = (float*)malloc(nnz*nnx*sizeof(float));
    d = (float*)malloc(nnz*nnx*sizeof(float));
    readFile(FNvelocity,FNepsilon,FNdelta,
             nx,nz,nnx,nnz,dx,dz,favg,dt,v,e,d,npml);

    /* alloc host and device record memory */
    shot_Hos=(float*)malloc(nt*nx*sizeof(float));

    /* initialize device, default device=0; */
    hipSetDevice(0);
    check_gpu_error("Failed to initialize device!");
    CHECK_gpu(hipDeviceReset());

    /* malloc the device media memory */
    hipMalloc(&vp, nnz*nnx*sizeof(float));
    hipMalloc(&epsilon, nnz*nnx*sizeof(float));
    hipMalloc(&delta, nnz*nnx*sizeof(float));

    /* copy the media parameters host memory to device */
    hipMemcpy(vp, v, nnz*nnx*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(epsilon, e, nnz*nnx*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(delta, d, nnz*nnx*sizeof(float), hipMemcpyHostToDevice);

    /* source wavefield device memory */
    hipMalloc(&s_u0, nnz*nnx*sizeof(float)); hipMalloc(&s_u1, nnz*nnx*sizeof(float));
    hipMalloc(&s_w0, nnz*nnx*sizeof(float)); hipMalloc(&s_w1, nnz*nnx*sizeof(float));
    hipMalloc(&s_P, nnz*nnx*sizeof(float));  hipMalloc(&s_Q, nnz*nnx*sizeof(float));
    hipMalloc(&s_px0, nnz*nnx*sizeof(float));hipMalloc(&s_px1, nnz*nnx*sizeof(float));
    hipMalloc(&s_pz0, nnz*nnx*sizeof(float));hipMalloc(&s_pz1, nnz*nnx*sizeof(float));
    hipMalloc(&s_qx0, nnz*nnx*sizeof(float));hipMalloc(&s_qx1, nnz*nnx*sizeof(float));
    hipMalloc(&s_qz0, nnz*nnx*sizeof(float));hipMalloc(&s_qz1, nnz*nnx*sizeof(float));

    /* boundary absorb coefficient device memory */
    hipMalloc(&coffx1, nnx*sizeof(float));   hipMalloc(&acoffx1, nnx*sizeof(float));
    hipMalloc(&coffx2, nnx*sizeof(float));   hipMalloc(&acoffx2, nnx*sizeof(float));
    hipMalloc(&coffz1, nnz*sizeof(float));   hipMalloc(&acoffz1, nnz*sizeof(float));
    hipMalloc(&coffz2, nnz*sizeof(float));   hipMalloc(&acoffz2, nnz*sizeof(float));

    hipMalloc(&shot_Dev, nx*nt*sizeof(float));

    /* imaging device memory */
    hipMalloc(&illumination, nz*nx*sizeof(float));

    /* check Nvidia GPU */
    check_gpu_error("Failed to allocate memory for variables!");

    /* calculate d0 and pml adsorb coffe */
    get_d0<<<1, 1>>>(dx, dz, nnx, nnz, npml, vp);
    initial_coffe<<<(nnx+511)/512, 512>>>(dt,nx,coffx1,coffx2,acoffx1,acoffx2,npml);
    initial_coffe<<<(nnz+511)/512, 512>>>(dt,nz,coffz1,coffz2,acoffz1,acoffz2,npml);

    /* set Imaging to zero */
    hipMemset(illumination, 0, nz*nx*sizeof(float));

    clock_t time;

    char msg[1024];
    /* Starting IS loop */
    for(is=1; is<=ns; is++)	 {

        time = clock();
        sprintf(msg, "IS =%3d/%d ",is,ns);
        print_info(msg);
        mBar(1.0*is/(1.0*ns));

        hipMemset(s_u0, 0, nnz*nnx*sizeof(float));  hipMemset(s_u1, 0, nnz*nnx*sizeof(float));
        hipMemset(s_w0, 0, nnz*nnx*sizeof(float));  hipMemset(s_w1, 0, nnz*nnx*sizeof(float));
        hipMemset(s_P, 0, nnz*nnx*sizeof(float));   hipMemset(s_Q, 0, nnz*nnx*sizeof(float));
        hipMemset(s_px0, 0, nnz*nnx*sizeof(float)); hipMemset(s_px1, 0, nnz*nnx*sizeof(float));
        hipMemset(s_pz0, 0, nnz*nnx*sizeof(float)); hipMemset(s_pz1, 0, nnz*nnx*sizeof(float));
        hipMemset(s_qx0, 0, nnz*nnx*sizeof(float)); hipMemset(s_qx1, 0, nnz*nnx*sizeof(float));
        hipMemset(s_qz0, 0, nnz*nnx*sizeof(float)); hipMemset(s_qz1, 0, nnz*nnx*sizeof(float));

        hipMemset(shot_Dev, 0, nt*nx*sizeof(float));

        /* forward */
        for(it=0,t=dt; it<nt; it++,t+=dt) {

            add_source<<<1,1>>>(pfac, fs,zs,nx,nz,nnx,nnz,dt,t,favg,wtype,npml,is,ds,s_P,s_Q);
            update_vel<<<(nnx*nnz+511)/512, 512>>>
                        (nx,nz,nnx,nnz,npml,dt,dx,dz,
                         s_u0,s_w0,s_u1,s_w1,s_P,s_Q,coffx1,coffx2,coffz1,coffz2);
            update_stress<<<(nnx*nnz+511)/512, 512>>>
                          (nx,nz,nnx,nnz,dt,dx,dz,s_u1,s_w1,s_P,s_Q,vp,npml,
                           s_px1,s_px0,s_pz1,s_pz0,s_qx1,s_qx0,s_qz1,s_qz0,
                           acoffx1,acoffx2,acoffz1,acoffz2,delta,epsilon,fs,ds,zs,is,false);
            s_u0 = s_u1;   s_w0 = s_w1;
            s_px0 = s_px1; s_pz0 = s_pz1;
            s_qx0 = s_qx1; s_qz0 = s_qz1;

            shot_record<<<(nx+511)/512, 512>>>
                        (nnx, nnz, nx, nz, npml, it, nt, s_P, shot_Dev, true);
            cal_illumination<<<(nx*nz+511)/512, 512>>>
                        (nnx, nnz, nz, npml, illumination, s_P, s_Q);

            if(writeSnap && (it%300==0)) {
                hipMemcpy(e, s_P, nnz*nnx*sizeof(float), hipMemcpyDeviceToHost);
                for(int i = npml; i<nnx-npml; i++)
                    for(int j = npml; j<nnz-npml; j++)
                        fwrite(&e[i*nnz+j], 4L, 1, fpSnap);
            }

        }//it

        mute_directwave<<<(nt+511)/512, 512>>>
                        (nx,nt,dt,favg,dx,dz,fs,ds,zs,is,vp,epsilon,shot_Dev,100);

        hipMemcpy(shot_Hos, shot_Dev, nt*nx*sizeof(float), hipMemcpyDeviceToHost);
        fwrite(shot_Hos,sizeof(float),nt*nx,fpCalShot);

        time = clock() - time;
        printf(", %f min\n", ((float)time)/60.0/CLOCKS_PER_SEC);

    }//is

    /* output multi-shot illumination */
    hipMemcpy(e, illumination, nz*nx*sizeof(float), hipMemcpyDeviceToHost);
    fwrite(e,sizeof(float),nx*nz,fpIllunmination);


    /* file close */
    if(writeSnap)
        fclose(fpSnap);
    fclose(fpCalShot);
    fclose(fpIllunmination);

    /* device memory free */
    hipFree(coffx1);     hipFree(acoffx1);
    hipFree(coffx2);     hipFree(acoffx2);
    hipFree(coffz1);     hipFree(acoffz1);
    hipFree(coffz2);     hipFree(acoffz2);

    hipFree(s_u0);       hipFree(s_u1);
    hipFree(s_w0);       hipFree(s_w1);
    hipFree(s_P);        hipFree(s_Q);
    hipFree(s_px0);      hipFree(s_px1);
    hipFree(s_pz0);      hipFree(s_pz1);
    hipFree(s_qx0);      hipFree(s_qx1);
    hipFree(s_qz0);      hipFree(s_qz1);

    hipFree(shot_Dev);

    hipFree(illumination);

    /* host memory free */
    free(v);
    free(e);
    free(d);
    free(shot_Hos);

    print_success("Finish Finite Difference\n");
}//FD

/**
 * RTM
 *         Author: Rong Tao
 */
void RTM(char FNvelocity[],
         char FNepsilon[],
         char FNdelta[],
         char FNObsShot[],
         char FNCalShot[],
         char FNSnap[],
         char FNMigration[],
         char FNIllumination[],
         char FNAdcigs[],
         char FNStkAdcigs[],
         char FNIntervalAdcigs[],
         int wtype,
         int npml,
         int nx,
         int nz,
         float dx,
         float dz,
         int nt,
         float dt,
         int ns,
         int fs,
         int ds,
         int zs,
         float favg,
         float pfac,
         int nangle,
         int dangle,
         int dAdcigs,
         bool readShot,
         bool writeSnap)
{
    float *v, *e, *d;
    float *vp, *epsilon, *delta;

    float *s_u0, *s_u1, *s_px0, *s_qx0, *s_px1, *s_qx1;
    float *s_w0, *s_w1, *s_pz0, *s_qz0, *s_pz1, *s_qz1;
    float *g_u0, *g_u1, *g_px0, *g_qx0, *g_px1, *g_qx1;
    float *g_w0, *g_w1, *g_pz0, *g_qz0, *g_pz1, *g_qz1;

    float *s_P, *s_Q, *g_P, *g_Q;

    float *coffx1,*coffx2,*coffz1,*coffz2;
    float *acoffx1,*acoffx2,*acoffz1,*acoffz2;

    float *shot_Dev, *shot_Hos, *P_bndr, *Q_bndr;
    float *migration, *illumination, *adcigs;
    float *Atemp;

    int nnx, nnz;
    int it, is;

    float t;

    /* FILE pointer */
    FILE *fpObsShot, *fpCalShot;
    if(readShot) {

        if((fpObsShot = fopen(FNObsShot,"rb"))==NULL){
            print_error(" ");
            printf("error open <%s>!\n",FNObsShot);
            exit(0);
        }
    }else{

        fpCalShot = fopen(FNCalShot,"wb");
    }
    FILE *fpSnap;
    if(writeSnap) {

        fpSnap = fopen(FNSnap,"wb");
    }
    FILE *fpMigration         = fopen(FNMigration,"wb");
    FILE *fpIllunmination     = fopen(FNIllumination,"wb");
    FILE *fpAdcigs            = fopen(FNAdcigs,"wb");
    FILE *fpStkAdcigs         = fopen(FNStkAdcigs,"wb");
    FILE *fpIntervalAdcigs    = fopen(FNIntervalAdcigs,"wb");

    /* whole media size */
    nnx = nx + 2*npml;
    nnz = nz + 2*npml;

    /* temp malloc for host adcigs */
    Atemp = (float*)malloc(nz*nx*nangle*sizeof(float));

    /* read the media file into memory */
    v = (float*)malloc(nnz*nnx*sizeof(float));
    e = (float*)malloc(nnz*nnx*sizeof(float));
    d = (float*)malloc(nnz*nnx*sizeof(float));
    readFile(FNvelocity,FNepsilon,FNdelta,
             nx,nz,nnx,nnz,dx,dz,favg,dt,v,e,d,npml);

    /* alloc host and device record memory */
    shot_Hos=(float*)malloc(nt*nx*sizeof(float));

    /* initialize device, default device=0; */
    hipSetDevice(0);
    check_gpu_error("Failed to initialize device!");
    CHECK_gpu(hipDeviceReset());

    /* malloc the device media memory */
    hipMalloc(&vp, nnz*nnx*sizeof(float));
    hipMalloc(&epsilon, nnz*nnx*sizeof(float));
    hipMalloc(&delta, nnz*nnx*sizeof(float));

    /* copy the media parameters host memory to device */
    hipMemcpy(vp, v, nnz*nnx*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(epsilon, e, nnz*nnx*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(delta, d, nnz*nnx*sizeof(float), hipMemcpyHostToDevice);

    /* source wavefield device memory */      /* receiver wavefield device memory */
    hipMalloc(&s_u0, nnz*nnx*sizeof(float));  hipMalloc(&g_u0, nnz*nnx*sizeof(float));
    hipMalloc(&s_u1, nnz*nnx*sizeof(float));  hipMalloc(&g_u1, nnz*nnx*sizeof(float));
    hipMalloc(&s_w0, nnz*nnx*sizeof(float));  hipMalloc(&g_w0, nnz*nnx*sizeof(float));
    hipMalloc(&s_w1, nnz*nnx*sizeof(float));  hipMalloc(&g_w1, nnz*nnx*sizeof(float));

    hipMalloc(&s_P, nnz*nnx*sizeof(float));   hipMalloc(&g_P, nnz*nnx*sizeof(float));
    hipMalloc(&s_Q, nnz*nnx*sizeof(float));   hipMalloc(&g_Q, nnz*nnx*sizeof(float));

    hipMalloc(&s_px0, nnz*nnx*sizeof(float)); hipMalloc(&g_px0, nnz*nnx*sizeof(float));
    hipMalloc(&s_px1, nnz*nnx*sizeof(float)); hipMalloc(&g_px1, nnz*nnx*sizeof(float));
    hipMalloc(&s_pz0, nnz*nnx*sizeof(float)); hipMalloc(&g_pz0, nnz*nnx*sizeof(float));
    hipMalloc(&s_pz1, nnz*nnx*sizeof(float)); hipMalloc(&g_pz1, nnz*nnx*sizeof(float));
    hipMalloc(&s_qx0, nnz*nnx*sizeof(float)); hipMalloc(&g_qx0, nnz*nnx*sizeof(float));
    hipMalloc(&s_qx1, nnz*nnx*sizeof(float)); hipMalloc(&g_qx1, nnz*nnx*sizeof(float));
    hipMalloc(&s_qz0, nnz*nnx*sizeof(float)); hipMalloc(&g_qz0, nnz*nnx*sizeof(float));
    hipMalloc(&s_qz1, nnz*nnx*sizeof(float)); hipMalloc(&g_qz1, nnz*nnx*sizeof(float));

    /* boundary absorb coefficient device memory */
    hipMalloc(&coffx1, nnx*sizeof(float));    hipMalloc(&acoffx1, nnx*sizeof(float));
    hipMalloc(&coffx2, nnx*sizeof(float));    hipMalloc(&acoffx2, nnx*sizeof(float));
    hipMalloc(&coffz1, nnz*sizeof(float));    hipMalloc(&acoffz1, nnz*sizeof(float));
    hipMalloc(&coffz2, nnz*sizeof(float));    hipMalloc(&acoffz2, nnz*sizeof(float));

    /* boundary wavefield device memory */
    hipMalloc(&P_bndr, nt*(2*nx+2*nz)*sizeof(float));
    hipMalloc(&Q_bndr, nt*(2*nx+2*nz)*sizeof(float));

    hipMalloc(&shot_Dev, nx*nt*sizeof(float));

    /* imaging device memory */
    hipMalloc(&migration, nz*nx*sizeof(float));
    hipMalloc(&illumination, nz*nx*sizeof(float));
    hipMalloc(&adcigs, nz*nangle*nx*sizeof(float));

    /* check Nvidia GPU */
    check_gpu_error("Failed to allocate memory for variables!");

    /* calculate d0 and pml adsorb coffe */
    get_d0<<<1, 1>>>(dx, dz, nnx, nnz, npml, vp);
    initial_coffe<<<(nnx+511)/512, 512>>>(dt,nx,coffx1,coffx2,acoffx1,acoffx2,npml);
    initial_coffe<<<(nnz+511)/512, 512>>>(dt,nz,coffz1,coffz2,acoffz1,acoffz2,npml);

    /* set Imaging to zero */
    hipMemset(migration, 0, nz*nx*sizeof(float));
    hipMemset(illumination, 0, nz*nx*sizeof(float));
    hipMemset(adcigs, 0, nz*nangle*nx*sizeof(float));

    clock_t time;

    char msg[1024];
    /* Starting IS loop */
    for(is=1; is<=ns; is++)	 {

        time = clock();
        sprintf(msg, "IS =%3d/%d ",is,ns);
        print_info(msg);
        mBar(1.0*is/(1.0*ns));

        hipMemset(s_u0, 0, nnz*nnx*sizeof(float)); hipMemset(g_u0, 0, nnz*nnx*sizeof(float));
        hipMemset(s_u1, 0, nnz*nnx*sizeof(float)); hipMemset(g_u1, 0, nnz*nnx*sizeof(float));
        hipMemset(s_w0, 0, nnz*nnx*sizeof(float)); hipMemset(g_w0, 0, nnz*nnx*sizeof(float));
        hipMemset(s_w1, 0, nnz*nnx*sizeof(float)); hipMemset(g_w1, 0, nnz*nnx*sizeof(float));

        hipMemset(s_P, 0, nnz*nnx*sizeof(float));  hipMemset(g_P, 0, nnz*nnx*sizeof(float));
        hipMemset(s_Q, 0, nnz*nnx*sizeof(float));  hipMemset(g_Q, 0, nnz*nnx*sizeof(float));

        hipMemset(s_px0, 0, nnz*nnx*sizeof(float));hipMemset(g_px0, 0, nnz*nnx*sizeof(float));
        hipMemset(s_px1, 0, nnz*nnx*sizeof(float));hipMemset(g_px1, 0, nnz*nnx*sizeof(float));
        hipMemset(s_pz0, 0, nnz*nnx*sizeof(float));hipMemset(g_pz0, 0, nnz*nnx*sizeof(float));
        hipMemset(s_pz1, 0, nnz*nnx*sizeof(float));hipMemset(g_pz1, 0, nnz*nnx*sizeof(float));
        hipMemset(s_qx0, 0, nnz*nnx*sizeof(float));hipMemset(g_qx0, 0, nnz*nnx*sizeof(float));
        hipMemset(s_qx1, 0, nnz*nnx*sizeof(float));hipMemset(g_qx1, 0, nnz*nnx*sizeof(float));
        hipMemset(s_qz0, 0, nnz*nnx*sizeof(float));hipMemset(g_qz0, 0, nnz*nnx*sizeof(float));
        hipMemset(s_qz1, 0, nnz*nnx*sizeof(float));hipMemset(g_qz1, 0, nnz*nnx*sizeof(float));

        hipMemset(shot_Dev, 0, nt*nx*sizeof(float));
        hipMemset(P_bndr, 0, nt*(2*nx+2*nz)*sizeof(float));
        hipMemset(Q_bndr, 0, nt*(2*nx+2*nz)*sizeof(float));

        /* forward */
        for(it=0,t=dt; it<nt; it++,t+=dt) {

            add_source<<<1,1>>>(pfac, fs,zs,nx,nz,nnx,nnz,dt,t,favg,wtype,npml,is,ds,s_P,s_Q);
            update_vel<<<(nnx*nnz+511)/512, 512>>>
                        (nx,nz,nnx,nnz,npml,dt,dx,dz,
                         s_u0,s_w0,s_u1,s_w1,s_P,s_Q,coffx1,coffx2,coffz1,coffz2);
            update_stress<<<(nnx*nnz+511)/512, 512>>>
                          (nx,nz,nnx,nnz,dt,dx,dz,s_u1,s_w1,s_P,s_Q,vp,npml,
                           s_px1,s_px0,s_pz1,s_pz0,s_qx1,s_qx0,s_qz1,s_qz0,
                           acoffx1,acoffx2,acoffz1,acoffz2,delta,epsilon,fs,ds,zs,is,true);
            s_u0 = s_u1;   s_w0 = s_w1;
            s_px0 = s_px1; s_pz0 = s_pz1;
            s_qx0 = s_qx1; s_qz0 = s_qz1;

            shot_record<<<(nx+511)/512, 512>>>
                        (nnx, nnz, nx, nz, npml, it, nt, s_P, shot_Dev, true);
            wavefield_bndr<<<((2*nx+2*nz)+511)/512,512>>>
                        (nnx, nnz, nx, nz, npml, it, nt, s_P, s_Q, P_bndr, Q_bndr, true);
            cal_illumination<<<(nx*nz+511)/512, 512>>>
                        (nnx, nnz, nz, npml, illumination, s_P, s_Q);

            if(writeSnap && (it%300==0)) {
                hipMemcpy(e, s_P, nnz*nnx*sizeof(float), hipMemcpyDeviceToHost);
                for(int i = npml; i<nnx-npml; i++)
                    for(int j = npml; j<nnz-npml; j++)
                        fwrite(&e[i*nnz+j], 4L, 1, fpSnap);
            }

        }//it

        mute_directwave<<<(nt+511)/512, 512>>>
                        (nx,nt,dt,favg,dx,dz,fs,ds,zs,is,vp,epsilon,shot_Dev,100);

        if(readShot) {

            fread(shot_Hos,sizeof(float),nt*nx,fpObsShot);
            hipMemcpy(shot_Dev, shot_Hos, nt*nx*sizeof(float), hipMemcpyHostToDevice);

        } else {

            hipMemcpy(shot_Hos, shot_Dev, nt*nx*sizeof(float), hipMemcpyDeviceToHost);
            fwrite(shot_Hos,sizeof(float),nt*nx,fpCalShot);
        }

        /* backward */
        for(it=nt-1; it>=0; it--) {

            /* source wavefield */
            wavefield_bndr<<<((2*nx+2*nz)+511)/512,512>>>
                            (nnx, nnz, nx, nz, npml, it, nt, s_P, s_Q, P_bndr, Q_bndr, false);
            update_vel<<<(nnx*nnz+511)/512, 512>>>
                            (nx,nz,nnx,nnz,npml,dt,dx,dz,
                             s_u0,s_w0,s_u1,s_w1,s_P,s_Q,coffx1,coffx2,coffz1,coffz2);
            update_stress<<<(nnx*nnz+511)/512, 512>>>
                            (nx,nz,nnx,nnz,dt,dx,dz,s_u1,s_w1,s_P,s_Q,vp,npml,
                             s_px1,s_px0,s_pz1,s_pz0,s_qx1,s_qx0,s_qz1,s_qz0,
                             acoffx1,acoffx2,acoffz1,acoffz2,delta,epsilon,fs,ds,zs,is,false);
            s_u0=s_u1;   s_w0=s_w1;
            s_px0=s_px1; s_pz0=s_pz1;
            s_qx0=s_qx1; s_qz0=s_qz1;

            if(writeSnap && (it%300==0)) {
                hipMemcpy(e, s_P, nnz*nnx*sizeof(float), hipMemcpyDeviceToHost);
                for(int i=npml;i<nnx-npml;i++)
                    for(int j=npml;j<nnz-npml;j++)
                        fwrite(&e[i*nnz+j],4L,1,fpSnap);
            }



            /* receivers wavefield */
            shot_record<<<(nx+511)/512, 512>>>
                        (nnx, nnz, nx, nz, npml, it, nt, g_P, shot_Dev, false);
            shot_record<<<(nx+511)/512, 512>>>
                        (nnx, nnz, nx, nz, npml, it, nt, g_Q, shot_Dev, false);
            update_vel<<<(nnx*nnz+511)/512, 512>>>
                        (nx,nz,nnx,nnz,npml,dt,dx,dz,
                         g_u0,g_w0,g_u1,g_w1,g_P,g_Q,coffx1,coffx2,coffz1,coffz2);
            update_stress<<<(nnx*nnz+511)/512, 512>>>
                            (nx,nz,nnx,nnz,dt,dx,dz,g_u1,g_w1,g_P,g_Q,vp,npml,
                             g_px1,g_px0,g_pz1,g_pz0,g_qx1,g_qx0,g_qz1,g_qz0,
                             acoffx1,acoffx2,acoffz1,acoffz2,
                             delta,epsilon,fs,ds,zs,is,false);
            g_u0=g_u1;   g_w0=g_w1;
            g_px0=g_px1; g_pz0=g_pz1;
            g_qx0=g_qx1; g_qz0=g_qz1;

            if(writeSnap && (it%300==0)) {
                hipMemcpy(e, g_P, nnz*nnx*sizeof(float), hipMemcpyDeviceToHost);
                for(int i=npml;i<nnx-npml;i++)
                    for(int j=npml;j<nnz-npml;j++)
                        fwrite(&e[i*nnz+j],4L,1,fpSnap);
            }

            cal_migration<<<(nx*nz+511)/512, 512>>>
                            (nnx, nnz, nz, npml, migration, s_P, g_P);

            Poynting_Adcigs<<<(nx*nz+511)/512, 512>>>
                            (nnz, nx, nz, npml, nangle, dangle, adcigs,
                             s_P, s_Q, s_u0, s_w0, g_P, g_Q, g_u0, g_w0);

        }//it

        time = clock() - time;
        printf(", %f min\n", ((float)time)/60.0/CLOCKS_PER_SEC);

    }//is

    migration_illum<<<(nx*nz+511)/512, 512>>>(nx, nz, npml, migration, illumination);

    adcigs_illum<<<(nx*nz*nangle+511)/512, 512>>>(nx, nz, nangle, dangle, adcigs, illumination);

    /* output multi-shot migration */
    hipMemcpy(e, migration, nz*nx*sizeof(float), hipMemcpyDeviceToHost);
    laplace_filter(1,nz,nx,e,d);
    fwrite(d,sizeof(float),nx*nz,fpMigration);

    /* output multi-shot illumination */
    hipMemcpy(e, illumination, nz*nx*sizeof(float), hipMemcpyDeviceToHost);
    fwrite(e,sizeof(float),nx*nz,fpIllunmination);

    /* output multi-shot adcigs */
    hipMemcpy(Atemp, adcigs, nz*nx*nangle*sizeof(float), hipMemcpyDeviceToHost);
    fwrite(Atemp,sizeof(float),nz*nx*nangle,fpAdcigs);

    /* output adcigs stk migration */
    stk_adcigs(nx,nz,nangle,Atemp,d);
    fwrite(d,sizeof(float),nx*nz,fpStkAdcigs);

    /* output smiled adcigs */
    adcigs_smiled(nx,nz,nangle,dAdcigs,Atemp);
    fwrite(Atemp,sizeof(float),nz*nx/dAdcigs*nangle,fpIntervalAdcigs);


    /* file close */
    if(writeSnap)
        fclose(fpSnap);
    if(readShot)
        fclose(fpObsShot);
    else
        fclose(fpCalShot);
    fclose(fpMigration);
    fclose(fpIllunmination);
    fclose(fpAdcigs);
    fclose(fpStkAdcigs);
    fclose(fpIntervalAdcigs);

    /* device memory free */
    hipFree(coffx1);    hipFree(acoffx1);
    hipFree(coffx2);    hipFree(acoffx2);
    hipFree(coffz1);    hipFree(acoffz1);
    hipFree(coffz2);    hipFree(acoffz2);

    hipFree(s_u0);    hipFree(s_u1);
    hipFree(s_w0);    hipFree(s_w1);

    hipFree(s_P);    hipFree(s_Q);

    hipFree(s_px0);    hipFree(s_px1);
    hipFree(s_pz0);    hipFree(s_pz1);
    hipFree(s_qx0);    hipFree(s_qx1);
    hipFree(s_qz0);    hipFree(s_qz1);

    hipFree(g_u0);    hipFree(g_u1);
    hipFree(g_w0);    hipFree(g_w1);

    hipFree(g_P);    hipFree(g_Q);

    hipFree(g_px0);    hipFree(g_px1);
    hipFree(g_pz0);    hipFree(g_pz1);
    hipFree(g_qx0);    hipFree(g_qx1);
    hipFree(g_qz0);    hipFree(g_qz1);

    hipFree(shot_Dev);

    hipFree(P_bndr);    hipFree(Q_bndr);

    hipFree(migration);
    hipFree(illumination);
    hipFree(adcigs);

    /* host memory free */
    free(v);
    free(e);
    free(d);
    free(shot_Hos);
    free(Atemp);

    print_success("Finish Reverse Time Migration!\n");

}//RTM


/**
 *     MAIN FUNCTION
 *         Author: Rong Tao
 */
int main(int argc,char *argv[])
{
    char msg[1024];
    #pragma message("\n\nNote:                   \
                     \n./a.out kind=1 for FD,  \
                     \n./a.out kind=2 for RTM  \
                     \n")


    /* this "if" for arguments line: ./a.out 1 */
    int kind;
    if(argc == 1 ){
        /* clear screen */
        system("clear");
        print_warning("Check the infomation.\n");
        for(int i=0; note[i] != NULL; i++) {

            fprintf(stderr, "%s",note[i]);
            if(i>13) {
                char ch = getchar();
                if(ch == 'q'){
                    sprintf(msg, "%s exit %s: line %d \n",argv[0],__FILE__,__LINE__);
                    print_success(msg);
                    exit(0);
                } else
                    continue;
            } else {
                fprintf(stderr, "\n");
            }
        }
        //assert(argc-1);
        exit(0);

    }else if(argc >= 2){

        sscanf(argv[1],"kind=%d",&kind);

        if(kind != 1 && kind != 2) {
            sprintf(msg, "Let %s kind=1 for FD. =2 for RTM.\n",argv[0]);
            print_error(msg);
            exit(0);

        }else {

            sprintf(msg, "The %s Starting...\n", argv[0]);
            print_info(msg);
        }
    }

    /* Parameters */
    int nx, nz, nt, wtype, nangle, dangle, dAdcigs;
    int ns, ds, fs, zs, npml;
    float dx, dz, dt, pfac, favg;
    bool readShot, writeSnap;

    clock_t start, end;

    /* file */
    /* these for FD */
    char FNvelocity[90]       = {"waxian_vel_201201201.dat"};
    char FNepsilon[90]        = {"waxian_eps_201201201.dat"};
    char FNdelta[90]          = {"waxian_del_201201201.dat"};
    char FNCalShot[90]        = {"shot.dat"};//shot cal
    char FNSnap[90]           = {"snap.dat"};//snap
    char FNIllumination[90]   = {"illumination.dat"};

    /* these for RTM */
    char FNObsShot[90]        = {"shot_obs.dat"};//shot obs

    char FNMigration[90]      = {"migration.dat"};
    char FNAdcigs[90]         = {"adcigs.dat"};
    char FNStkAdcigs[90]      = {"stkadcigs.dat"};
    char FNIntervalAdcigs[90] = {"smiled_adcigs.dat"};


    wtype=1;/* wavelet: 1,2,3 */
    npml=20;/* pml boundary */

    readShot = false;/* true: read shot;
                       flase: use right shot record */
    writeSnap = false;/* true: write;
                        flase: no write snap */

    nx = 201;
    nz = 201;         favg=35;     pfac=1000.0;

    dx=10.0;
    dz=10.0;

    nt=2001;
    dt=0.001;

    ns=5;
    fs=nx/ns/2;
    ds=nx/ns;
    zs=1;

    nangle=70;
    dangle=1;
    dAdcigs=25;



    start = clock();


    if(kind == 1){
        /* FD */
        print_info("FD starting...\n");
        FD( FNvelocity,
            FNepsilon,
            FNdelta,
            FNCalShot,
            FNSnap,
            FNIllumination,
            wtype,
            npml,
            nx,
            nz,
            dx,
            dz,
            nt,
            dt,
            ns,
            fs,
            ds,
            zs,
            favg,
            pfac,
            writeSnap);

    }else if(kind == 2){
        /* RTM */
        print_info("RTM starting...\n");
        RTM(FNvelocity,
            FNepsilon,
            FNdelta,
            FNObsShot,
            FNCalShot,
            FNSnap,
            FNMigration,
            FNIllumination,
            FNAdcigs,
            FNStkAdcigs,
            FNIntervalAdcigs,
            wtype,
            npml,
            nx,
            nz,
            dx,
            dz,
            nt,
            dt,
            ns,
            fs,
            ds,
            zs,
            favg,
            pfac,
            nangle,
            dangle,
            dAdcigs,
            readShot,
            writeSnap);

    } else { }


    end = clock();

    sprintf(msg, "The %s Finished... \n", argv[0]);
    print_info(msg);

    sprintf(msg, "Total %d shots: %f (min)\n", ns, ((float)(end-start))/60.0/CLOCKS_PER_SEC);
    print_info(msg);
    print_success("Done！ File in:\n");
    system("pwd");

    return 0;

}//end of main