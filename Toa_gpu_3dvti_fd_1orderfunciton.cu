#include "hip/hip_runtime.h"
//a#########################################################
//a##         3D Acoustic VTI Medium Forward 
//a##    
//a##  Ps :GPU(CUDA)  
//a##	Copyright (C) Rong Tao, all rights reserved
//a##/*a***************************
//a##Function for VTI medium modeling,
//a##
//a## Ps:  the function of modeling following:
//a##      
//a##          du/dt=1/rho*dp/dx , 
//a##          dv/dt=1/rho*dp/dy , 
//a##          dw/dt=1/rho*dq/dz ,  
//a##          dp/dt=rho*vpx^2*(du/dx+dv/dy)+rho*vp*vpn*dw/dz ,
//a##          dq/dt=rho*vp*vpn*(du/dx+dv/dy)+rho*vp^2*dw/dz ,
//a##                     vpx^2=vp^2*(1+2*epsilon);
//a##                     vpn^2=vp^2*(1+2*delta);
//a##  
//a##*********a*******************/
//a##
//a##                                  code by Rong Tao 
//a##                            
//a#########################################################
#include <stdio.h>
#include <malloc.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define pi 3.141592653

#define BlockSize1 16// tile size in 1st-axis
#define BlockSize2 16// tile size in 2nd-axis

#define mm 4

__device__ float d0;

__constant__ float c[mm]={1.196289,-0.0797526,0.009570313,-0.0006975447};

//a################################################################################
void check_gpu_error (const char *msg) 
/*< check GPU errors >*/
{
    hipError_t err = hipGetLastError ();
    if (hipSuccess != err) { 
	printf("Cuda error: %s: %s\n", msg, hipGetErrorString (err)); 
	exit(0);   
    }
}
//a################################################################################
__global__ void add_source(float pfac,int fsx,int fsy,int sz,int nx,int ny,int nz,int nnx,int nny,int nnz,float dt,float t,
                        float favg,int wtype,int npml,int is,int dsx,int dsy,float *P,float *Q,int nsx)
/*< generate ricker wavelet with time deley >*/
{
       int ixs,iys,izs;
       float x_,xx_,tdelay,ts,source=0.0,sx,sy; 
  
       tdelay=1.0/favg;
       ts=t-tdelay;

       sx=fsx+is%nsx*dsx;
       sy=fsy+is/nsx*dsy;

	if(wtype==1)//ricker wavelet
	{
          x_=favg*ts;
          xx_=x_*x_;
          source=(1-2*pi*pi*(xx_))*exp(-(pi*pi*xx_));
	}else if(wtype==2){//derivative of gaussian
          x_=(-4)*favg*favg*pi*pi/log(0.1);
          source=(-2)*pi*pi*ts*exp(-x_*ts*ts);
        }else if(wtype==3){//derivative of gaussian
          x_=(-1)*favg*favg*pi*pi/log(0.1);
          source=exp(-x_*ts*ts);
        }

       if(t<=2*tdelay)
       {         
	     ixs = sx+npml-1;
	     iys = sy+npml-1;
            izs = sz+npml-1;
            P[izs+ixs*nnz+iys*nnz*nnx]+=pfac*source;
            Q[izs+ixs*nnz+iys*nnz*nnx]+=pfac*source;
       }
}
/*******************func*********************/
__global__ void update_vel(int nx,int ny,int nz,int nnx,int nny,int nnz,int npml,float dt,float dx,float dy,float dz,
                           float *u0,float *v0,float *w0,float *u1,float *v1,float *w1,float *P,float *Q,
                           float *coffx1,float *coffx2,float *coffy1,float *coffy2,float *coffz1,float *coffz2)
{
    const int iz = blockIdx.x * blockDim.x + threadIdx.x;//0--nz's thread:iz
    const int ix = blockIdx.y * blockDim.y + threadIdx.y;//0--nx's thread:ix

       int id,iy,im;
	float dtx,dty,dtz,xx,yy,zz;

		 dtx=dt/dx;
		 dty=dt/dy;
		 dtz=dt/dz;

       for(iy=0;iy<nny;iy++)
        {
               id=iz+ix*nnz+iy*nnz*nnx;
               if(id>=mm&&id<nnx*nny*nnz-mm)
                 {
                   if(ix>=mm&&ix<(nnx-mm)&&iy>=mm&&iy<(nny-mm)&&iz>=mm&&iz<(nnz-mm))
                    {
                     xx=0.0;
                     yy=0.0;
                     zz=0.0;
	             for(im=0;im<mm;im++)
                      {
                        yy+=c[im]*(P[id+(im+1)*nnz*nnx] - P[id-im*nnz*nnx]);
                        xx+=c[im]*(P[id+(im+1)*nnz]     - P[id-im*nnz]);
                        zz+=c[im]*(Q[id+im+1]           - Q[id-im]);
                      }
                     u1[id]=coffx2[ix]*u0[id]-coffx1[ix]*dtx*xx;
                     v1[id]=coffy2[iy]*v0[id]-coffy1[iy]*dty*yy;
                     w1[id]=coffz2[iz]*w0[id]-coffz1[iz]*dtz*zz;
                   }
                 }
        }  



}
/*******************func***********************/
__global__ void update_stress(int nx,int ny,int nz,int nnx,int nny,int nnz,float dt,float dx,float dy,float dz,
                           float *u1,float *v1,float *w1,float *P,float *Q,float *vp,int npml,
                           float *px1,float *px0,float *py1,float *py0,float *pz1,float *pz0,
                           float *qx1,float *qx0,float *qy1,float *qy0,float *qz1,float *qz0,
                           float *acoffx1,float *acoffx2,float *acoffy1,float *acoffy2,float *acoffz1,float *acoffz2,
                           float *delta,float *epsilon,int fsx,int dsx,int fsy,int dsy,int zs,int is,int nsx,bool SV)
{
    const int iz = blockIdx.x * blockDim.x + threadIdx.x;//0--nz's thread:iz
    const int ix = blockIdx.y * blockDim.y + threadIdx.y;//0--nx's thread:ix

       int id,iy,im,rx,ry,rz,R=15,r=4;
	float dtx,dty,dtz,xx,yy,zz,ee,dd;

		 dtx=dt/dx;
		 dty=dt/dy;
		 dtz=dt/dz;

       for(iy=0;iy<nny;iy++)
        {
               id=iz+ix*nnz+iy*nnz*nnx;
               if(id>=mm&&id<nnx*nnz*nny-mm)
                 {
/************************i****************************************/
/************************iso circle start*************************/
                   rx=ix-(fsx+is%nsx*dsx+npml-1);
                   ry=iy-(fsy+is/nsx*dsy+npml-1);
                   rz=iz-(zs+npml-1);
                   if(SV){
                       if((rx*rx+ry*ry+rz*rz)<=R*R){
                           if((rx*rx+ry*ry+rz*rz)<=r*r){
                               ee = 0.0;
                               dd = 0.0;
                           }else{
                               ee = 0.5*(1-cos(pi*((sqrtf(rx*rx+ry*ry+rz*rz)-r)*4.0/(R*3.0-1))))*epsilon[id];
                               dd = 0.5*(1-cos(pi*((sqrtf(rx*rx+ry*ry+rz*rz)-r)*4.0/(R*3.0-1))))*delta[id]; 
                              }
                       }else{
                          ee=epsilon[id];
                          dd=delta[id];
                          }
                   }else{
                      ee=epsilon[id];
                      dd=delta[id];
                     }
/************************ iso circle end *************************/
/************************i****************************************/
                   if(ix>=mm&&ix<(nnx-mm)&&iy>=mm&&iy<(nny-mm)&&iz>=mm&&iz<(nnz-mm))
                     {
                     xx=0.0;
                     yy=0.0;
                     zz=0.0;
	             for(im=0;im<mm;im++)
                       {
                        yy+=c[im]*(v1[id+im*nnz*nnx] - v1[id-(im+1)*nnz*nnx]);
                        xx+=c[im]*(u1[id+im*nnz]     - u1[id-(im+1)*nnz]);
                        zz+=c[im]*(w1[id+im]         - w1[id-im-1]);
                       }
                     px1[id]=acoffx2[ix]*px0[id] - acoffx1[ix]*vp[id]*vp[id]*(1+2*ee)*dtx*xx;
                     py1[id]=acoffy2[iy]*py0[id] - acoffy1[iy]*vp[id]*vp[id]*(1+2*ee)*dty*yy;
                     pz1[id]=acoffz2[iz]*pz0[id] - acoffz1[iz]*vp[id]*vp[id]*sqrtf(1+2*dd)*dtz*zz;

                     qx1[id]=acoffx2[ix]*qx0[id] - acoffx1[ix]*vp[id]*vp[id]*sqrtf(1+2*dd)*dtx*xx;
                     qy1[id]=acoffy2[iy]*qy0[id] - acoffy1[iy]*vp[id]*vp[id]*sqrtf(1+2*dd)*dty*yy;
                     qz1[id]=acoffz2[iz]*qz0[id] - acoffz1[iz]*vp[id]*vp[id]*dtz*zz;

                     P[id]=px1[id]+py1[id]+pz1[id];
                     Q[id]=qx1[id]+qy1[id]+qz1[id];
                   }
                 }
         }
}                      
/********************func**********************/
__global__ void get_d0(float dx,float dy,float dz,int nnx,int nny,int nnz,int npml,float *vp)
{
   d0=10.0*vp[nny*nnx*nnz/2]*log(100000.0)/(2.0*npml*((dx+dy+dz)/3.0));
}
/*************func*******************/
void pad_vv(int nx,int ny,int nz,int nnx,int nny,int nnz,int npml,float *ee)
{
     int ix,iy,iz,id;
 
	    for(iy=0;iy<nny;iy++)
		 for(ix=0;ix<nnx;ix++)
		 {
			 for(iz=0;iz<nnz;iz++)
			 {
                             id=iz+ix*nnz+iy*nnz*nnx;

                             if(ix<npml){
                                ee[id]=ee[iz+npml*nnz+iy*nnz*nnx];  //left
                             }else if(ix>=nnx-npml){
                                ee[id]=ee[iz+(nnx-npml-1)*nnz+iy*nnz*nnx];//right
                                 }
			 }
		 }
	    for(iy=0;iy<nny;iy++)
		 for(ix=0;ix<nnx;ix++)
		 {
			 for(iz=0;iz<nnz;iz++)
			 {
                             id=iz+ix*nnz+iy*nnz*nnx;

                             if(iy<npml){
                                ee[id]=ee[iz+ix*nnz+npml*nnz*nnx];  //front
                             }else if(iy>=nny-npml){
                                ee[id]=ee[iz+ix*nnz+(nny-npml-1)*nnz*nnx];//back
                                 }
			 }
		 }
	    for(iy=0;iy<nny;iy++)
		 for(ix=0;ix<nnx;ix++)
		 {
			 for(iz=0;iz<nnz;iz++)
			 {
                             id=iz+ix*nnz+iy*nnz*nnx;

                             if(iz<npml){
                                ee[id]=ee[npml+ix*nnz+iy*nnz*nnx];  //up
                             }else if(iz>=nnz-npml){
                                ee[id]=ee[nnz-npml-1+ix*nnz+iy*nnz*nnx];//down
                                 }
			 }
		 }

}
/*************func*******************/
void read_file(char FN1[],char FN2[],char FN3[],int nx,int ny,int nz,int nnx,int nny,int nnz,float *vv,float *epsilon,float *delta,int npml)
{
		 int ix,iy,iz,id;
		
		 FILE *fp1,*fp2,*fp3;
		 if((fp1=fopen(FN1,"rb"))==NULL)printf("error open <%s>!\n",FN1);
		 if((fp2=fopen(FN2,"rb"))==NULL)printf("error open <%s>!\n",FN2);
		 if((fp3=fopen(FN3,"rb"))==NULL)printf("error open <%s>!\n",FN3);

	    for(iy=npml;iy<ny+npml;iy++)
		 for(ix=npml;ix<nx+npml;ix++)
		 {
			 for(iz=npml;iz<nz+npml;iz++)
			 {
                             id=iz+ix*nnz+iy*nnz*nnx;
				 fread(&vv[id],4L,1,fp1);//vv[id]=3000.0;
				 fread(&epsilon[id],4L,1,fp2);epsilon[id] *= 0.0;//epsilon[id] = (epsilon[id]-3000)/800.0*0.25;
				 fread(&delta[id],4L,1,fp3);delta[id] *= 0.0;//delta[id] = (delta[id]-3000)/800.0*0.1;
			 }
		 }
		 fclose(fp1);
		 fclose(fp2);
		 fclose(fp3);
}
/*************func*******************/
__global__ void initial_coffe(float dt,int nn,float *coff1,float *coff2,float *acoff1,float *acoff2,int npml)
{		
	 int id=threadIdx.x+blockDim.x*blockIdx.x;

           if(id<nn+2*npml)
            {
		 if(id<npml)
		 {   
			 coff1[id]=1.0/(1.0+(dt*d0*pow((npml-0.5-id)/npml,2.0))/2.0);
			 coff2[id]=coff1[id]*(1.0-(dt*d0*pow((npml-0.5-id)/npml,2.0))/2.0);

			 acoff1[id]=1.0/(1.0+(dt*d0*pow(((npml-id)*1.0)/npml,2.0))/2.0);
			 acoff2[id]=acoff1[id]*(1.0-(dt*d0*pow(((npml-id)*1.0)/npml,2.0))/2.0);

		 }else if(id>=npml&&id<npml+nn){

			 coff1[id]=1.0;
			 coff2[id]=1.0;

			 acoff1[id]=1.0;
			 acoff2[id]=1.0;

		 }else{

			 coff1[id]=1.0/(1.0+(dt*d0*pow((0.5+id-nn-npml)/npml,2.0))/2.0);
			 coff2[id]=coff1[id]*(1.0-(dt*d0*pow((0.5+id-nn-npml)/npml,2.0))/2.0);

			 acoff1[id]=1.0/(1.0+(dt*d0*pow(((id-nn-npml)*1.0)/npml,2.0))/2.0);
			 acoff2[id]=acoff1[id]*(1.0-(dt*d0*pow(((id-nn-npml)*1.0)/npml,2.0))/2.0);
		 }	
            }       
}
/*************func*******************/
__global__ void shot_record(int nnx,int nny, int nnz, int nx,int ny, int nz, int npml, int it, int nt, float *P, float *shot)
{		
	 int id=threadIdx.x+blockDim.x*blockIdx.x;

        int ix=id%nx;
        int iy=id/nx;

           if(id<nx*ny)
            {
               shot[it+nt*ix+nt*nx*iy]=P[npml+nnz*(ix+npml)+nnz*nnx*(iy+npml)];
            }       
}

/*************func**************/ 
void window3d(float *a, float *b, int nz, int nx, int ny, int nnz, int nnx, int npml)
/*< window a 3d subvolume >*/
{
	int iz, ix, iy;
	
	for(iy=0; iy<ny; iy++)
	for(ix=0; ix<nx; ix++)
	for(iz=0; iz<nz; iz++)
	{
		a[iz+nz*ix+nz*nx*iy]=b[(iz+npml)+nnz*(ix+npml)+nnz*nnx*(iy+npml)];
	}
}
/*************func**************/    
__global__ void mute_directwave(int nx,int ny,int nt,float dt,float favg, float dx,float dy,float dz,int fsx,int fsy,int dsx,int dsy,
                                int zs,int is, float *vp,float *epsilon,float *shot,int tt,int nsx)
{

    const int ix = blockIdx.x * blockDim.x + threadIdx.x;
    const int iy = blockIdx.y * blockDim.y + threadIdx.y;

    int id,it;
    int mu_t,mu_nt;
    float mu_x,mu_y,mu_z,mu_t0;

       for(it=0;it<nt;it++)
        {
          id=it+ix*nt+iy*nx*nt;
          if(ix<nx&&iy<ny&&it<nt)
            {
              mu_x=dx*abs(ix-fsx-(is%nsx)*dsx);
              mu_y=dy*abs(iy-fsy-(is/nsx)*dsy);
              mu_z=dz*zs;
              mu_t0=sqrtf(pow(mu_x,2)+pow(mu_y,2)+pow(mu_z,2))/(vp[1]*sqrtf(1+2*epsilon[1]));
              mu_t=(int)(2.0/(dt*favg));
              mu_nt=(int)(mu_t0/dt)+mu_t+tt;

                 if(it<mu_nt)
                    shot[id]=0.0;
            }
        }
/*    int id=threadIdx.x+blockDim.x*blockIdx.x;

    int mu_t,mu_nt;
    float mu_x,mu_y,mu_z,mu_t0;

    int ix=(id/nt)%nx;
    int iy=(id/nt)/nx;
    int it=id%nt;

   if(id<nx*ny*nt)
   {
        mu_x=dx*abs(ix-fsx-(is%nsx)*dsx);
        mu_y=dy*abs(iy-fsy-(is/nsx)*dsy);
        mu_z=dz*zs;
        mu_t0=sqrtf(pow(mu_x,2)+pow(mu_y,2)+pow(mu_z,2))/(vp[1]*sqrtf(1+2*epsilon[1]));
        mu_t=(int)(2.0/(dt*favg));
        mu_nt=(int)(mu_t0/dt)+mu_t+tt;

           if(it<mu_nt)
              shot[id]=0.0;
   }  */
}
//a########################################################################
int main(int argc,char *argv[])
{
	int is, it, nx, ny, nz, nnx, nny, nnz, nt, wtype;
	int ns, nsx, dsx, fsx, dsy, fsy, zs, npml;
	float dx, dy, dz, dt, t, pfac, favg;

	float *v, *e, *d;
	float *vp, *epsilon, *delta;
	float *s_u0, *s_u1, *s_px0, *s_qx0, *s_px1, *s_qx1;
	float *s_v0, *s_v1, *s_py0, *s_qy0, *s_py1, *s_qy1;
       float *s_w0, *s_w1, *s_pz0, *s_qz0, *s_pz1, *s_qz1;
	float *s_P, *s_Q, *shot_Dev, *shot_Hos;

       float *coffx1,*coffx2,*coffy1,*coffy2,*coffz1,*coffz2;
       float *acoffx1,*acoffx2,*acoffy1,*acoffy2,*acoffz1,*acoffz2;

       clock_t start, end;
/*************wavelet\boundary**************/
          wtype=1;npml=20;
/********** dat document ***********/
          char FN1[250]={"waxian_vel_301301201.dat"};
          char FN2[250]={"waxian_eps_301301201.dat"};
          char FN3[250]={"waxian_del_301301201.dat"};
	   char FN4[250]={"waxian_shot_1501_iso_mute.dat"};
	   char FN5[250]={"waxian_snap.dat"};

/********aaa************/  
	 FILE *fpsnap, *fpshot;
        fpshot=fopen(FN4,"wb");
        fpsnap=fopen(FN5,"wb");

 
/********* parameters *************/

          nx=301; 
          ny=301;              
	   nz=201;         favg=30;     pfac=10.0;

 	   dx=10.0;  
 	   dy=10.0;   
        dz=10.0;   
     
          nt=1501;    
          dt=0.001;
     
          ns=1;          nsx=1;  
          fsx=nx/nsx/2;    
          dsx=nx/nsx;         
          fsy=ny/(ns/nsx)/2;   
          dsy=ny/(ns/nsx);
          zs=1;     
/*************v***************/ 
          nnx=nx+2*npml;
          nny=ny+2*npml;
          nnz=nz+2*npml;
/************a*************/


    	 v=(float*)malloc(nnz*nnx*nny*sizeof(float));
    	 e=(float*)malloc(nnz*nnx*nny*sizeof(float));
    	 d=(float*)malloc(nnz*nnx*nny*sizeof(float));
    	 shot_Hos=(float*)malloc(nt*nx*ny*sizeof(float));
        read_file(FN1,FN2,FN3,nx,ny,nz,nnx,nny,nnz,v,e,d,npml);
/****************************/
        pad_vv(nx,ny,nz,nnx,nny,nnz,npml,e);
        pad_vv(nx,ny,nz,nnx,nny,nnz,npml,d);
        pad_vv(nx,ny,nz,nnx,nny,nnz,npml,v);

        hipSetDevice(0);// initialize device, default device=0;
	 check_gpu_error("Failed to initialize device!");

	dim3 Xdimg, dimg, dimb;
	Xdimg.x=(nnx+BlockSize1-1)/BlockSize1;
	Xdimg.y=(nny+BlockSize2-1)/BlockSize2;
	dimg.x=(nnz+BlockSize1-1)/BlockSize1;
	dimg.y=(nnx+BlockSize2-1)/BlockSize2;
	dimb.x=BlockSize1;
	dimb.y=BlockSize2;
/****************************/
        hipMalloc(&vp, nnz*nnx*nny*sizeof(float));
        hipMalloc(&epsilon, nnz*nnx*nny*sizeof(float));
        hipMalloc(&delta, nnz*nnx*nny*sizeof(float));
	 hipMemcpy(vp, v, nnz*nnx*nny*sizeof(float), hipMemcpyHostToDevice);
	 hipMemcpy(epsilon, e, nnz*nnx*nny*sizeof(float), hipMemcpyHostToDevice);
	 hipMemcpy(delta, d, nnz*nnx*nny*sizeof(float), hipMemcpyHostToDevice);
/****************************/
        hipMalloc(&s_u0, nnz*nnx*nny*sizeof(float));    hipMalloc(&s_u1, nnz*nnx*nny*sizeof(float));
        hipMalloc(&s_v0, nnz*nnx*nny*sizeof(float));    hipMalloc(&s_v1, nnz*nnx*nny*sizeof(float));
        hipMalloc(&s_w0, nnz*nnx*nny*sizeof(float));    hipMalloc(&s_w1, nnz*nnx*nny*sizeof(float));

        hipMalloc(&s_P, nnz*nnx*nny*sizeof(float));     hipMalloc(&s_Q, nnz*nnx*nny*sizeof(float));

        hipMalloc(&s_px0, nnz*nnx*nny*sizeof(float));   hipMalloc(&s_px1, nnz*nnx*nny*sizeof(float));
        hipMalloc(&s_py0, nnz*nnx*nny*sizeof(float));   hipMalloc(&s_py1, nnz*nnx*nny*sizeof(float));
        hipMalloc(&s_pz0, nnz*nnx*nny*sizeof(float));   hipMalloc(&s_pz1, nnz*nnx*nny*sizeof(float));
        hipMalloc(&s_qx0, nnz*nnx*nny*sizeof(float));   hipMalloc(&s_qx1, nnz*nnx*nny*sizeof(float));
        hipMalloc(&s_qy0, nnz*nnx*nny*sizeof(float));   hipMalloc(&s_qy1, nnz*nnx*nny*sizeof(float));
        hipMalloc(&s_qz0, nnz*nnx*nny*sizeof(float));   hipMalloc(&s_qz1, nnz*nnx*nny*sizeof(float));

        hipMalloc(&coffx1, nnx*sizeof(float));     hipMalloc(&coffx2, nnx*sizeof(float));
        hipMalloc(&coffy1, nnx*sizeof(float));     hipMalloc(&coffy2, nnx*sizeof(float));
        hipMalloc(&coffz1, nnz*sizeof(float));     hipMalloc(&coffz2, nnz*sizeof(float));
        hipMalloc(&acoffx1, nnx*sizeof(float));    hipMalloc(&acoffx2, nnx*sizeof(float));
        hipMalloc(&acoffy1, nnx*sizeof(float));    hipMalloc(&acoffy2, nnx*sizeof(float));
        hipMalloc(&acoffz1, nnz*sizeof(float));    hipMalloc(&acoffz2, nnz*sizeof(float));

        hipMalloc(&shot_Dev, nx*ny*nt*sizeof(float));
/******************************/
	 check_gpu_error("Failed to allocate memory for variables!");

        get_d0<<<1, 1>>>(dx,dy,dz,nnx,nny,nnz,npml,vp);
        initial_coffe<<<(nnx+511)/512, 512>>>(dt,nx,coffx1,coffx2,acoffx1,acoffx2,npml);
        initial_coffe<<<(nny+511)/512, 512>>>(dt,ny,coffy1,coffy2,acoffy1,acoffy2,npml);
        initial_coffe<<<(nnz+511)/512, 512>>>(dt,nz,coffz1,coffz2,acoffz1,acoffz2,npml);



        printf("--------------------------------------------------------\n");
        printf("---   \n");   
        start = clock();                                  
/**********IS Loop start*******/
   for(is=0;is<ns;is++)	
    {     
       //  printf("---   IS=%3d  \n",is);

     hipMemset(s_u0, 0, nnz*nnx*nny*sizeof(float));     hipMemset(s_u1, 0, nnz*nnx*nny*sizeof(float));
     hipMemset(s_v0, 0, nnz*nnx*nny*sizeof(float));     hipMemset(s_v1, 0, nnz*nnx*nny*sizeof(float));
     hipMemset(s_w0, 0, nnz*nnx*nny*sizeof(float));     hipMemset(s_w1, 0, nnz*nnx*nny*sizeof(float));

     hipMemset(s_P, 0, nnz*nnx*nny*sizeof(float));      hipMemset(s_Q, 0, nnz*nnx*nny*sizeof(float));

     hipMemset(s_px0, 0, nnz*nnx*nny*sizeof(float));    hipMemset(s_px1, 0, nnz*nnx*nny*sizeof(float));
     hipMemset(s_py0, 0, nnz*nnx*nny*sizeof(float));    hipMemset(s_py1, 0, nnz*nnx*nny*sizeof(float));
     hipMemset(s_pz0, 0, nnz*nnx*nny*sizeof(float));    hipMemset(s_pz1, 0, nnz*nnx*nny*sizeof(float));
     hipMemset(s_qx0, 0, nnz*nnx*nny*sizeof(float));    hipMemset(s_qx1, 0, nnz*nnx*nny*sizeof(float));
     hipMemset(s_qy0, 0, nnz*nnx*nny*sizeof(float));    hipMemset(s_qy1, 0, nnz*nnx*nny*sizeof(float));
     hipMemset(s_qz0, 0, nnz*nnx*nny*sizeof(float));    hipMemset(s_qz1, 0, nnz*nnx*nny*sizeof(float));

     hipMemset(shot_Dev, 0, nt*nx*ny*sizeof(float));

     for(it=0,t=dt;it<nt;it++,t+=dt)
     { 
      if(it%100==0)printf("---   IS===%d   it===%d\n",is,it);
        add_source<<<1,1>>>(pfac,fsx,fsy,zs,nx,ny,nz,nnx,nny,nnz,dt,t,favg,wtype,npml,is,dsx,dsy,s_P,s_Q,nsx);
        update_vel<<<dimg,dimb>>>(nx,ny,nz,nnx,nny,nnz,npml,dt,dx,dy,dz,
                                 s_u0,s_v0,s_w0,s_u1,s_v1,s_w1,s_P,s_Q,coffx1,coffx2,coffy1,coffy2,coffz1,coffz2);
        update_stress<<<dimg,dimb>>>(nx,ny,nz,nnx,nny,nnz,dt,dx,dy,dz,s_u1,s_v1,s_w1,s_P,s_Q,vp,npml,
                                     s_px1,s_px0,s_py1,s_py0,s_pz1,s_pz0,s_qx1,s_qx0,s_qy1,s_qy0,s_qz1,s_qz0,
                                     acoffx1,acoffx2,acoffy1,acoffy2,acoffz1,acoffz2,delta,epsilon, 
                                     fsx, dsx, fsy, dsy,zs, is, nsx, true);
        s_u0=s_u1; s_v0=s_v1; s_w0=s_w1; s_px0=s_px1; s_py0=s_py1; s_pz0=s_pz1; s_qx0=s_qx1; s_qy0=s_qy1; s_qz0=s_qz1; 

        shot_record<<<(nx*ny+511)/512, 512>>>(nnx,nny, nnz, nx,ny, nz, npml, it, nt, s_P, shot_Dev);


           if((is==1)&&(it!=0&&it%300==0))
            {
	       hipMemcpy(e, s_P, nnz*nnx*nny*sizeof(float), hipMemcpyDeviceToHost);
              fseek(fpsnap,(int)(it/300-1)*(nx*ny*nz)*4L,0);
              window3d(v, e, nz, nx, ny, nnz, nnx, npml);
              fwrite(v,4L,nx*nz*ny,fpsnap);
            }
     }//it loop end
      mute_directwave<<<Xdimg,dimb>>>(nx,ny,nt,dt,favg,dx,dy,dz,fsx,fsy,dsx,dsy,zs,is,vp,epsilon,shot_Dev,60,nsx);
      hipMemcpy(shot_Hos, shot_Dev, nt*nx*ny*sizeof(float), hipMemcpyDeviceToHost);
      fseek(fpshot,is*nt*nx*ny*sizeof(float),0);
      fwrite(shot_Hos,sizeof(float),nt*nx*ny,fpshot);

    }//is loop end
    end = clock();
/*********IS Loop end*********/ 		     
   printf("---   The forward is over    \n"); 
   printf("---   Complete!!!!!!!!! \n");  
   printf("total %d shots: %f (s)\n", ns, ((float)(end-start))/CLOCKS_PER_SEC);



/***********close************/ 
          fclose(fpsnap);   fclose(fpshot);
/***********free*************/ 
       hipFree(coffx1);       hipFree(coffx2);
       hipFree(coffz1);       hipFree(coffz2);
       hipFree(acoffx1);      hipFree(acoffx2);
       hipFree(acoffz1);      hipFree(acoffz2);

       hipFree(s_u0);           hipFree(s_u1);
       hipFree(s_v0);           hipFree(s_v1);
       hipFree(s_w0);           hipFree(s_w1);

       hipFree(s_P);            hipFree(s_Q);

       hipFree(s_px0);          hipFree(s_px1);
       hipFree(s_py0);          hipFree(s_py1);
       hipFree(s_pz0);          hipFree(s_pz1);
       hipFree(s_qx0);          hipFree(s_qx1);
       hipFree(s_qy0);          hipFree(s_qy1);
       hipFree(s_qz0);          hipFree(s_qz1);

       hipFree(shot_Dev);
/***************host free*****************/
	free(v);	free(e);	free(d);
       free(shot_Hos);
}

