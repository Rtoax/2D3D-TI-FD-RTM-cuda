#include "hip/hip_runtime.h"
//a#########################################################
//a##         3D Acoustic VTI Medium RTM
//a##
//a##  Ps :GPU(CUDA)  ,-SV    ,3D laplace filter
//a##
//a##/*a***************************
//a##Function for VTI medium modeling,
//a##
//a## Ps:  the function of modeling following:
//a##
//a##          du/dt=1/rho*dp/dx ,
//a##          dv/dt=1/rho*dp/dy ,
//a##          dw/dt=1/rho*dq/dz ,
//a##          dp/dt=rho*vpx^2*(du/dx+dv/dy)+rho*vp*vpn*dw/dz ,
//a##          dq/dt=rho*vp*vpn*(du/dx+dv/dy)+rho*vp^2*dw/dz ,
//a##                     vpx^2=vp^2*(1+2*epsilu);
//a##                     vpn^2=vp^2*(1+2*deta);
//a##
//a##*********a*******************/
//a##	Copyright (C) Rong Tao, all rights reserved
//a##                                   code by Rong Tao
//a##
//a#########################################################
#include <stdio.h>
#include <malloc.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define pi 3.141592653

#define BlockSize1 16// tile size in 1st-axis
#define BlockSize2 16// tile size in 2nd-axis

#define mm 4

__device__ float d0;

__constant__ float c[mm]={1.196289,-0.0797526,0.009570313,-0.0006975447};

//a################################################################################
void check_gpu_error (const char *msg)
/*< check GPU errors >*/
{
    hipError_t err = hipGetLastError ();
    if (hipSuccess != err) {
	printf("Cuda error: %s: %s\n", msg, hipGetErrorString (err));
	exit(0);
    }
}
//a################################################################################
__global__ void add_source(float pfac,int fsx,int fsy,int sz,int nx,int ny,int nz,int nnx,int nny,int nnz,float dt,float t,
                        float favg,int wtype,int npml,int is,int dsx,int dsy,float *P,float *Q,int nsx)
/*< generate ricker wavelet with time deley >*/
{
       int ixs,iys,izs;
       float x_,xx_,tdelay,ts,source=0.0,sx,sy;

       tdelay=1.0/favg;
       ts=t-tdelay;

      // sx=fsx+is%nsx*dsx;
      // sy=fsy+is/nsx*dsy;

       sx=fsx+is*dsx;
       sy=fsy+is*dsy;

	if(wtype==1)//ricker wavelet
	{
          x_=favg*ts;
          xx_=x_*x_;
          source=(1-2*pi*pi*(xx_))*exp(-(pi*pi*xx_));
	}else if(wtype==2){//derivative of gaussian
          x_=(-4)*favg*favg*pi*pi/log(0.1);
          source=(-2)*pi*pi*ts*exp(-x_*ts*ts);
        }else if(wtype==3){//derivative of gaussian
          x_=(-1)*favg*favg*pi*pi/log(0.1);
          source=exp(-x_*ts*ts);
        }

       if(t<=2*tdelay)
       {
	     ixs = sx+npml-1;
	     iys = sy+npml-1;
            izs = sz+npml-1;
            P[izs+ixs*nnz+iys*nnz*nnx]+=pfac*source;
            Q[izs+ixs*nnz+iys*nnz*nnx]+=pfac*source;
       }
}
/*******************func*********************/
__global__ void update_vel(int nx,int ny,int nz,int nnx,int nny,int nnz,int npml,float dt,float dx,float dy,float dz,
                           float *u0,float *v0,float *w0,float *u1,float *v1,float *w1,float *P,float *Q,
                           float *coffx1,float *coffx2,float *coffy1,float *coffy2,float *coffz1,float *coffz2)
{
    const int iz = blockIdx.x * blockDim.x + threadIdx.x;//0--nz's thread:iz
    const int ix = blockIdx.y * blockDim.y + threadIdx.y;//0--nx's thread:ix

       int id,iy,im;
	float dtx,dty,dtz,xx,yy,zz;

		 dtx=dt/dx;
		 dty=dt/dy;
		 dtz=dt/dz;

       for(iy=0;iy<nny;iy++)
        {
               id=iz+ix*nnz+iy*nnz*nnx;
               if(id>=mm&&id<nnx*nny*nnz-mm)
                 {
                   if(ix>=mm&&ix<(nnx-mm)&&iy>=mm&&iy<(nny-mm)&&iz>=mm&&iz<(nnz-mm))
                    {
                     xx=0.0;
                     yy=0.0;
                     zz=0.0;
	             for(im=0;im<mm;im++)
                      {
                        yy+=c[im]*(P[id+(im+1)*nnz*nnx] - P[id-im*nnz*nnx]);
                        xx+=c[im]*(P[id+(im+1)*nnz]     - P[id-im*nnz]);
                        zz+=c[im]*(Q[id+im+1]           - Q[id-im]);
                      }
                     u1[id]=coffx2[ix]*u0[id]-coffx1[ix]*dtx*xx;
                     v1[id]=coffy2[iy]*v0[id]-coffy1[iy]*dty*yy;
                     w1[id]=coffz2[iz]*w0[id]-coffz1[iz]*dtz*zz;
                   }
                 }
        }



}
/*******************func***********************/
__global__ void update_stress(int nx,int ny,int nz,int nnx,int nny,int nnz,float dt,float dx,float dy,float dz,
                           float *u1,float *v1,float *w1,float *P,float *Q,float *vp,int npml,
                           float *px1,float *px0,float *py1,float *py0,float *pz1,float *pz0,
                           float *qx1,float *qx0,float *qy1,float *qy0,float *qz1,float *qz0,
                           float *acoffx1,float *acoffx2,float *acoffy1,float *acoffy2,float *acoffz1,float *acoffz2,
                           float *deta,float *epsilu,int fsx,int dsx,int fsy,int dsy,int zs,int is,int nsx,bool SV)
{
    const int iz = blockIdx.x * blockDim.x + threadIdx.x;//0--nz's thread:iz
    const int ix = blockIdx.y * blockDim.y + threadIdx.y;//0--nx's thread:ix

       int id,iy,im,rx,ry,rz,R=15,r=4;
	float dtx,dty,dtz,xx,yy,zz,ee,dd;

		 dtx=dt/dx;
		 dty=dt/dy;
		 dtz=dt/dz;

       for(iy=0;iy<nny;iy++)
        {
               id=iz+ix*nnz+iy*nnz*nnx;
               if(id>=mm&&id<nnx*nnz*nny-mm)
                 {
/************************i****************************************/
/************************iso circle start*************************/
                  // rx=ix-(fsx+is%nsx*dsx+npml-1);
                  // ry=iy-(fsy+is/nsx*dsy+npml-1);
                   rx=ix-(fsx+is*dsx+npml-1);
                   ry=iy-(fsy+is*dsy+npml-1);
                   rz=iz-(zs+npml-1);
                   if(SV){
                       if((rx*rx+ry*ry+rz*rz)<=R*R){
                           if((rx*rx+ry*ry+rz*rz)<=r*r){
                               ee = 0.0;
                               dd = 0.0;
                           }else{
                               ee = 0.5*(1-cos(pi*((sqrtf(rx*rx+ry*ry+rz*rz)-r)*4.0/(R*3.0-1))))*epsilu[id];
                               dd = 0.5*(1-cos(pi*((sqrtf(rx*rx+ry*ry+rz*rz)-r)*4.0/(R*3.0-1))))*deta[id];
                              }
                       }else{
                          ee=epsilu[id];
                          dd=deta[id];
                          }
                   }else{
                      ee=epsilu[id];
                      dd=deta[id];
                     }
/************************ iso circle end *************************/
/************************i****************************************/
                   if(ix>=mm&&ix<(nnx-mm)&&iy>=mm&&iy<(nny-mm)&&iz>=mm&&iz<(nnz-mm))
                     {
                     xx=0.0;
                     yy=0.0;
                     zz=0.0;
	             for(im=0;im<mm;im++)
                       {
                        yy+=c[im]*(v1[id+im*nnz*nnx] - v1[id-(im+1)*nnz*nnx]);
                        xx+=c[im]*(u1[id+im*nnz]     - u1[id-(im+1)*nnz]);
                        zz+=c[im]*(w1[id+im]         - w1[id-im-1]);
                       }
                     px1[id]=acoffx2[ix]*px0[id] - acoffx1[ix]*vp[id]*vp[id]*(1+2*ee)*dtx*xx;
                     py1[id]=acoffy2[iy]*py0[id] - acoffy1[iy]*vp[id]*vp[id]*(1+2*ee)*dty*yy;
                     pz1[id]=acoffz2[iz]*pz0[id] - acoffz1[iz]*vp[id]*vp[id]*sqrtf(1+2*dd)*dtz*zz;

                     qx1[id]=acoffx2[ix]*qx0[id] - acoffx1[ix]*vp[id]*vp[id]*sqrtf(1+2*dd)*dtx*xx;
                     qy1[id]=acoffy2[iy]*qy0[id] - acoffy1[iy]*vp[id]*vp[id]*sqrtf(1+2*dd)*dty*yy;
                     qz1[id]=acoffz2[iz]*qz0[id] - acoffz1[iz]*vp[id]*vp[id]*dtz*zz;

                     P[id]=px1[id]+py1[id]+pz1[id];
                     Q[id]=qx1[id]+qy1[id]+qz1[id];
                   }
                 }
         }
}
/********************func**********************/
__global__ void get_d0(float dx,float dy,float dz,int nnx,int nny,int nnz,int npml,float *vp)
{
   d0=10.0*vp[nny*nnx*nnz/2]*log(100000.0)/(2.0*npml*((dx+dy+dz)/3.0));
}
/*************func*******************/
void pad_vv(int nx,int ny,int nz,int nnx,int nny,int nnz,int npml,float *ee)
{
     int ix,iy,iz,id;

	    for(iy=0;iy<nny;iy++)
		 for(ix=0;ix<nnx;ix++)
		 {
			 for(iz=0;iz<nnz;iz++)
			 {
                             id=iz+ix*nnz+iy*nnz*nnx;

                             if(ix<npml){
                                ee[id]=ee[iz+npml*nnz+iy*nnz*nnx];  //left
                             }else if(ix>=nnx-npml){
                                ee[id]=ee[iz+(nnx-npml-1)*nnz+iy*nnz*nnx];//right
                                 }
			 }
		 }
	    for(iy=0;iy<nny;iy++)
		 for(ix=0;ix<nnx;ix++)
		 {
			 for(iz=0;iz<nnz;iz++)
			 {
                             id=iz+ix*nnz+iy*nnz*nnx;

                             if(iy<npml){
                                ee[id]=ee[iz+ix*nnz+npml*nnz*nnx];  //front
                             }else if(iy>=nny-npml){
                                ee[id]=ee[iz+ix*nnz+(nny-npml-1)*nnz*nnx];//back
                                 }
			 }
		 }
	    for(iy=0;iy<nny;iy++)
		 for(ix=0;ix<nnx;ix++)
		 {
			 for(iz=0;iz<nnz;iz++)
			 {
                             id=iz+ix*nnz+iy*nnz*nnx;

                             if(iz<npml){
                                ee[id]=ee[npml+ix*nnz+iy*nnz*nnx];  //up
                             }else if(iz>=nnz-npml){
                                ee[id]=ee[nnz-npml-1+ix*nnz+iy*nnz*nnx];//down
                                 }
			 }
		 }

}
/*************func*******************/
void read_file(char FN1[],char FN2[],char FN3[],int nx,int ny,int nz,int nnx,int nny,int nnz,float *vv,float *epsilu,float *deta,int npml)
{
		 int ix,iy,iz,id;

		 FILE *fp1,*fp2,*fp3;
		 if((fp1=fopen(FN1,"rb"))==NULL){printf("error open <%s>!\n",FN1);exit(0);}
		 if((fp2=fopen(FN2,"rb"))==NULL){printf("error open <%s>!\n",FN2);exit(0);}
		 if((fp3=fopen(FN3,"rb"))==NULL){printf("error open <%s>!\n",FN3);exit(0);}

	    for(iy=npml;iy<ny+npml;iy++)
		 for(ix=npml;ix<nx+npml;ix++)
		 {
			 for(iz=npml;iz<nz+npml;iz++)
			 {
                             id=iz+ix*nnz+iy*nnz*nnx;
				 fread(&vv[id],4L,1,fp1);
				 fread(&epsilu[id],4L,1,fp2);
				 fread(&deta[id],4L,1,fp3);
			 }
		 }
		 fclose(fp1);
		 fclose(fp2);
		 fclose(fp3);
}
/*************func*******************/
__global__ void initial_coffe(float dt,int nn,float *coff1,float *coff2,float *acoff1,float *acoff2,int npml)
{
	 int id=threadIdx.x+blockDim.x*blockIdx.x;

           if(id<nn+2*npml)
            {
		 if(id<npml)
		 {
			 coff1[id]=1.0/(1.0+(dt*d0*pow((npml-0.5-id)/npml,2.0))/2.0);
			 coff2[id]=coff1[id]*(1.0-(dt*d0*pow((npml-0.5-id)/npml,2.0))/2.0);

			 acoff1[id]=1.0/(1.0+(dt*d0*pow(((npml-id)*1.0)/npml,2.0))/2.0);
			 acoff2[id]=acoff1[id]*(1.0-(dt*d0*pow(((npml-id)*1.0)/npml,2.0))/2.0);

		 }else if(id>=npml&&id<npml+nn){

			 coff1[id]=1.0;
			 coff2[id]=1.0;

			 acoff1[id]=1.0;
			 acoff2[id]=1.0;

		 }else{

			 coff1[id]=1.0/(1.0+(dt*d0*pow((0.5+id-nn-npml)/npml,2.0))/2.0);
			 coff2[id]=coff1[id]*(1.0-(dt*d0*pow((0.5+id-nn-npml)/npml,2.0))/2.0);

			 acoff1[id]=1.0/(1.0+(dt*d0*pow(((id-nn-npml)*1.0)/npml,2.0))/2.0);
			 acoff2[id]=acoff1[id]*(1.0-(dt*d0*pow(((id-nn-npml)*1.0)/npml,2.0))/2.0);
		 }
            }
}
/*************func*******************/
__global__ void shot_record(int nnx,int nny, int nnz, int nx,int ny, int nz, int npml, int it, int nt,
                            float *P, float *Q, float *shot, bool flag)
{
	 int id=threadIdx.x+blockDim.x*blockIdx.x;

        int ix=id%nx;
        int iy=id/nx;

           if(id<nx*ny)
            {
              if(flag)
                {
                 shot[it+nt*ix+nt*nx*iy]=P[npml+nnz*(ix+npml)+nnz*nnx*(iy+npml)]
                                        +Q[npml+nnz*(ix+npml)+nnz*nnx*(iy+npml)];
              }else{
                 P[npml-1+nnz*(ix+npml)+nnz*nnx*(iy+npml)]=shot[it+nt*ix+nt*nx*iy];
                 Q[npml-1+nnz*(ix+npml)+nnz*nnx*(iy+npml)]=shot[it+nt*ix+nt*nx*iy];
                }
            }
}

/*************func**************/
void window3d(float *a, float *b, int nz, int nx, int ny, int nnz, int nnx, int npml)
/*< window a 3d subvolume >*/
{
	int iz, ix, iy;

	for(iy=0; iy<ny; iy++)
	for(ix=0; ix<nx; ix++)
	for(iz=0; iz<nz; iz++)
	{
		a[iz+nz*ix+nz*nx*iy]=b[(iz+npml)+nnz*(ix+npml)+nnz*nnx*(iy+npml)];
	}
}
/*************func**************/
__global__ void mute_directwave(int nx,int ny,int nt,float dt,float favg, float dx,float dy,float dz,int fsx,int fsy,int dsx,int dsy,
                                int zs,int is, float *vp,float *epsilu,float *shot,int tt,int nsx)
{

    const int ix = blockIdx.x * blockDim.x + threadIdx.x;
    const int iy = blockIdx.y * blockDim.y + threadIdx.y;

    int id,it;
    int mu_t,mu_nt;
    float mu_x,mu_y,mu_z,mu_t0;

       for(it=0;it<nt;it++)
        {
          id=it+ix*nt+iy*nx*nt;
          if(ix<nx&&iy<ny&&it<nt)
            {
            //  mu_x=dx*abs(ix-fsx-(is%nsx)*dsx);
            //  mu_y=dy*abs(iy-fsy-(is/nsx)*dsy);
              mu_x=dx*abs(ix-fsx-is*dsx);
              mu_y=dy*abs(iy-fsy-is*dsy);
              mu_z=dz*zs;
              mu_t0=sqrtf(pow(mu_x,2)+pow(mu_y,2)+pow(mu_z,2))/(vp[1]*sqrtf(1+2*epsilu[1]));
              mu_t=(int)(2.0/(dt*favg));
              mu_nt=(int)(mu_t0/dt)+mu_t+tt;

           if((it>(int)(mu_t0/dt)-tt)&&(it<mu_nt))
                    shot[id]=0.0;
            }
        }
/*    int id=threadIdx.x+blockDim.x*blockIdx.x;

    int mu_t,mu_nt;
    float mu_x,mu_y,mu_z,mu_t0;

    int ix=(id/nt)%nx;
    int iy=(id/nt)/nx;
    int it=id%nt;

   if(id<nx*ny*nt)
   {
        mu_x=dx*abs(ix-fsx-(is%nsx)*dsx);
        mu_y=dy*abs(iy-fsy-(is/nsx)*dsy);
        mu_z=dz*zs;
        mu_t0=sqrtf(pow(mu_x,2)+pow(mu_y,2)+pow(mu_z,2))/(vp[1]*sqrtf(1+2*epsilu[1]));
        mu_t=(int)(2.0/(dt*favg));
        mu_nt=(int)(mu_t0/dt)+mu_t+tt;

           if(it<mu_nt)
              shot[id]=0.0;
   }  */
}
/*************func*******************/
__global__ void wavefield_bndr(int nnx, int nny, int nnz, int nx, int ny, int nz, int npml, int it, int nt,
                               float *P, float *Q, float *P_bndr, float *Q_bndr, bool flag)
{
	 int id=threadIdx.x+blockDim.x*blockIdx.x;
        int ix,iy,iz;

           if(id<2*nx*ny+2*nz*ny+2*nx*nz)
            {
            if(flag)/////////////////////////////////save boundary
             {
              if(id<nx*ny){//up

               ix=id%nx;
               iy=id/nx;
               P_bndr[it*(2*nx*ny+2*nz*ny+2*nx*nz)+id]=P[npml-1+nnz*(ix+npml)+nnz*nnx*(iy+npml)];
               Q_bndr[it*(2*nx*ny+2*nz*ny+2*nx*nz)+id]=Q[npml-1+nnz*(ix+npml)+nnz*nnx*(iy+npml)];

              }else if(id>=nx*ny&&id<(2*nx*ny)){//down

               ix=(id-nx*ny)%nx;
               iy=(id-nx*ny)/nx;
               P_bndr[it*(2*nx*ny+2*nz*ny+2*nx*nz)+id]=P[npml+nz+nnz*(ix+npml)+nnz*nnx*(iy+npml)];
               Q_bndr[it*(2*nx*ny+2*nz*ny+2*nx*nz)+id]=Q[npml+nz+nnz*(ix+npml)+nnz*nnx*(iy+npml)];

              }else if(id>=(2*nx*ny)&&id<(2*nx*ny+nz*ny)){//left

               iz=(id-2*nx*ny)%nz;
               iy=(id-2*nx*ny)/nz;
               P_bndr[it*(2*nx*ny+2*nz*ny+2*nx*nz)+id]=P[npml+iz+nnz*(npml-1)+nnz*nnx*(iy+npml)];
               Q_bndr[it*(2*nx*ny+2*nz*ny+2*nx*nz)+id]=Q[npml+iz+nnz*(npml-1)+nnz*nnx*(iy+npml)];

              }else if(id>=(2*nx*ny+nz*ny)&&id<(2*nx*ny+2*nz*ny)){//right

               iz=(id-2*nx*ny-nz*ny)%nz;
               iy=(id-2*nx*ny-nz*ny)/nz;
               P_bndr[it*(2*nx*ny+2*nz*ny+2*nx*nz)+id]=P[npml+iz+nnz*(nx+npml)+nnz*nnx*(iy+npml)];
               Q_bndr[it*(2*nx*ny+2*nz*ny+2*nx*nz)+id]=Q[npml+iz+nnz*(nx+npml)+nnz*nnx*(iy+npml)];

              }else if(id>=(2*nx*ny+2*nz*ny)&&id<(2*nx*ny+2*nz*ny+nx*nz)){//front

               iz=(id-2*nx*ny-2*nz*ny)%nz;
               ix=(id-2*nx*ny-2*nz*ny)/nz;
               P_bndr[it*(2*nx*ny+2*nz*ny+2*nx*nz)+id]=P[npml+iz+nnz*(ix+npml)+nnz*nnx*(npml-1)];
               Q_bndr[it*(2*nx*ny+2*nz*ny+2*nx*nz)+id]=Q[npml+iz+nnz*(ix+npml)+nnz*nnx*(npml-1)];

                }else if(id>=(2*nx*ny+2*nz*ny+nx*nz)&&id<(2*nx*ny+2*nz*ny+2*nx*nz)){//back

               iz=(id-2*nx*ny-2*nz*ny-nx*nz)%nz;
               ix=(id-2*nx*ny-2*nz*ny-nx*nz)/nz;
               P_bndr[it*(2*nx*ny+2*nz*ny+2*nx*nz)+id]=P[npml+iz+nnz*(ix+npml)+nnz*nnx*(npml+ny)];
               Q_bndr[it*(2*nx*ny+2*nz*ny+2*nx*nz)+id]=Q[npml+iz+nnz*(ix+npml)+nnz*nnx*(npml+ny)];

                }

             }else{

              if(id<nx*ny){//up

               ix=id%nx;
               iy=id/nx;
               P[npml-1+nnz*(ix+npml)+nnz*nnx*(iy+npml)]=P_bndr[it*(2*nx*ny+2*nz*ny+2*nx*nz)+id];
               Q[npml-1+nnz*(ix+npml)+nnz*nnx*(iy+npml)]=Q_bndr[it*(2*nx*ny+2*nz*ny+2*nx*nz)+id];

              }else if(id>=nx*ny&&id<(2*nx*ny)){//down

               ix=(id-nx*ny)%nx;
               iy=(id-nx*ny)/nx;
               P[npml+nz+nnz*(ix+npml)+nnz*nnx*(iy+npml)]=P_bndr[it*(2*nx*ny+2*nz*ny+2*nx*nz)+id];
               Q[npml+nz+nnz*(ix+npml)+nnz*nnx*(iy+npml)]=Q_bndr[it*(2*nx*ny+2*nz*ny+2*nx*nz)+id];

              }else if(id>=(2*nx*ny)&&id<(2*nx*ny+nz*ny)){//left

               iz=(id-2*nx*ny)%nz;
               iy=(id-2*nx*ny)/nz;
               P[npml+iz+nnz*(npml-1)+nnz*nnx*(iy+npml)]=P_bndr[it*(2*nx*ny+2*nz*ny+2*nx*nz)+id];
               Q[npml+iz+nnz*(npml-1)+nnz*nnx*(iy+npml)]=Q_bndr[it*(2*nx*ny+2*nz*ny+2*nx*nz)+id];

              }else if(id>=(2*nx*ny+nz*ny)&&id<(2*nx*ny+2*nz*ny)){//right

               iz=(id-2*nx*ny-nz*ny)%nz;
               iy=(id-2*nx*ny-nz*ny)/nz;
               P[npml+iz+nnz*(nx+npml)+nnz*nnx*(iy+npml)]=P_bndr[it*(2*nx*ny+2*nz*ny+2*nx*nz)+id];
               Q[npml+iz+nnz*(nx+npml)+nnz*nnx*(iy+npml)]=Q_bndr[it*(2*nx*ny+2*nz*ny+2*nx*nz)+id];

              }else if(id>=(2*nx*ny+2*nz*ny)&&id<(2*nx*ny+2*nz*ny+nx*nz)){//front

               iz=(id-2*nx*ny-2*nz*ny)%nz;
               ix=(id-2*nx*ny-2*nz*ny)/nz;
               P[npml+iz+nnz*(ix+npml)+nnz*nnx*(npml-1)]=P_bndr[it*(2*nx*ny+2*nz*ny+2*nx*nz)+id];
               Q[npml+iz+nnz*(ix+npml)+nnz*nnx*(npml-1)]=Q_bndr[it*(2*nx*ny+2*nz*ny+2*nx*nz)+id];

                }else if(id>=(2*nx*ny+2*nz*ny+nx*nz)&&id<(2*nx*ny+2*nz*ny+2*nx*nz)){//back

               iz=(id-2*nx*ny-2*nz*ny-nx*nz)%nz;
               ix=(id-2*nx*ny-2*nz*ny-nx*nz)/nz;
               P[npml+iz+nnz*(ix+npml)+nnz*nnx*(npml+ny)]=P_bndr[it*(2*nx*ny+2*nz*ny+2*nx*nz)+id];
               Q[npml+iz+nnz*(ix+npml)+nnz*nnx*(npml+ny)]=Q_bndr[it*(2*nx*ny+2*nz*ny+2*nx*nz)+id];

                }
             }
            }
}
/*************func**************/
__global__ void cal_migration(int nnx, int nny, int nnz,int nx, int ny, int nz, int npml, float *migration, float *s, float *g)
{
    const int iz = blockIdx.x * blockDim.x + threadIdx.x;//0--nz's thread:iz
    const int ix = blockIdx.y * blockDim.y + threadIdx.y;//0--nx's thread:ix

       int id,iy;

       for(iy=0;iy<ny;iy++)
        {
           id=iz+ix*nz+iy*nz*nx;
           if(ix<nx&&iy<ny&&iz<nz)
              migration[id]+=s[iz+npml+nnz*(ix+npml)+nnx*nnz*(iy+npml)]*g[iz+npml+nnz*(ix+npml)+nnx*nnz*(iy+npml)];
        }
}
/*************func**************/
__global__ void cal_illumination(int nnx, int nny, int nnz,int nx, int ny, int nz, int npml, float *illumination, float *P, float *Q)
{
    const int iz = blockIdx.x * blockDim.x + threadIdx.x;//0--nz's thread:iz
    const int ix = blockIdx.y * blockDim.y + threadIdx.y;//0--nx's thread:ix

       int id,iy;

       for(iy=0;iy<ny;iy++)
        {
           id=iz+ix*nz+iy*nz*nx;
           if(ix<nx&&iy<ny&&iz<nz)
              illumination[id]+=P[iz+npml+nnz*(ix+npml)+nnx*nnz*(iy+npml)]*P[iz+npml+nnz*(ix+npml)+nnx*nnz*(iy+npml)]
                               +Q[iz+npml+nnz*(ix+npml)+nnx*nnz*(iy+npml)]*Q[iz+npml+nnz*(ix+npml)+nnx*nnz*(iy+npml)];
        }
}
/*************func**************/
__global__ void migration_illum(int nnx, int nny, int nnz,int nx, int ny, int nz, int npml, float *illumination, float *migration)
{
    const int iz = blockIdx.x * blockDim.x + threadIdx.x;//0--nz's thread:iz
    const int ix = blockIdx.y * blockDim.y + threadIdx.y;//0--nx's thread:ix

       int id,iy;

       for(iy=0;iy<ny;iy++)
        {
           id=iz+ix*nz+iy*nz*nx;
           if(ix<nx&&iy<ny&&iz<nz)
               if(illumination[id]!=0)
                     migration[id]/=illumination[id];
        }
}
/*************func**************/
void laplace_3d_filter(int adj, int nz, int nx,int ny, float *in, float *out)
/*< linear operator, come from Madagascar Mlaplac2>*/
{
    int iz,ix,iy,j;
    for (j=0;j<nx*nz*ny;j++) out[j]=0.0;

  for(iy=0;iy<ny;iy++)
    for (ix=0; ix < nx; ix++) {
	for (iz=0; iz < nz; iz++) {

	    j = iz+ix*nz+iy*nx*nz;
	    if (iz > 0) {
		if (adj) {
		    out[j-1] -= in[j];
		    out[j]   += in[j];
		} else {
		    out[j] += in[j] - in[j-1];
		}
	    }
	    if (iz < nz-1) {
		if (adj) {
		    out[j+1] -= in[j];
		    out[j]   += in[j];
		} else {
		    out[j] += in[j] - in[j+1];
		}
	    }
	    if (ix > 0) {
		if (adj) {
		    out[j-nz] -= in[j];
		    out[j]    += in[j];
		} else {
		    out[j] += in[j] - in[j-nz];
		}
	    }
	    if (ix < nx-1) {
		if (adj) {
		    out[j+nz] -= in[j];
		    out[j]    += in[j];
		} else {
		    out[j] += in[j] - in[j+nz];
		}
	    }
	    if (iy > 0) {
		if (adj) {
		    out[j-nz*nx] -= in[j];
		    out[j]    += in[j];
		} else {
		    out[j] += in[j] - in[j-nz*nx];
		}
	    }
	    if (iy < ny-1) {
		if (adj) {
		    out[j+nz*nx] -= in[j];
		    out[j]    += in[j];
		} else {
		    out[j] += in[j] - in[j+nz*nx];
		}
	    }
	}
    }
}
/*************func**************/
__global__ void Poynting_Adcigs(int nnx, int nny, int nnz, int nx, int ny, int nz, int npml, int na, int da,float *adcigs,int dcdp,
                           float *s_P, float *s_Q, float *s_u, float *s_v, float *s_w,
                           float *g_P, float *g_Q, float *g_u, float *g_v, float *g_w)
{
    const int iz = blockIdx.x * blockDim.x + threadIdx.x;//0--nz's thread:iz
    const int ix = blockIdx.y * blockDim.y + threadIdx.y;//0--nx's thread:ix

       int id,iy,ia;

    float Ssx, Ssy, Ssz, Sgx, Sgy, Sgz, b1, b2, a;

       for(iy=0;iy<(int)(ny/dcdp);iy++)
        {

           if(ix<(int)(nx/dcdp)&&iz<nz)
            {
               ia=0;
               Ssx=-s_P[iz+npml+nnz*(ix*dcdp+npml)+nnz*nnx*(iy*dcdp+npml)]*s_u[iz+npml+nnz*(ix*dcdp+npml)+nnz*nnx*(iy*dcdp+npml)];
               Ssy=-s_P[iz+npml+nnz*(ix*dcdp+npml)+nnz*nnx*(iy*dcdp+npml)]*s_v[iz+npml+nnz*(ix*dcdp+npml)+nnz*nnx*(iy*dcdp+npml)];
               Ssz=-s_Q[iz+npml+nnz*(ix*dcdp+npml)+nnz*nnx*(iy*dcdp+npml)]*s_w[iz+npml+nnz*(ix*dcdp+npml)+nnz*nnx*(iy*dcdp+npml)];
               Sgx= g_P[iz+npml+nnz*(ix*dcdp+npml)+nnz*nnx*(iy*dcdp+npml)]*g_u[iz+npml+nnz*(ix*dcdp+npml)+nnz*nnx*(iy*dcdp+npml)];
               Sgy= g_P[iz+npml+nnz*(ix*dcdp+npml)+nnz*nnx*(iy*dcdp+npml)]*g_v[iz+npml+nnz*(ix*dcdp+npml)+nnz*nnx*(iy*dcdp+npml)];
               Sgz= g_Q[iz+npml+nnz*(ix*dcdp+npml)+nnz*nnx*(iy*dcdp+npml)]*g_w[iz+npml+nnz*(ix*dcdp+npml)+nnz*nnx*(iy*dcdp+npml)];

               b1= Ssx*Ssx + Ssy*Ssy + Ssz*Ssz;
               b2= Sgx*Sgx + Sgy*Sgy + Sgz*Sgz;
                a=(Ssx*Sgx + Ssy*Sgy + Ssz*Sgz)/(sqrtf(b1*b2)*(1 - 0.1));

               if(a>=-1&&a<=1)
                 {
                   a=0.5*acosf(a)*180.0/pi;
                   ia=(int)(a/(da*1.0));
                   if(ia<na)
                     {
                        id=iz+ia*nz+ix*na*nz+iy*nz*na*((int)(nx/dcdp));
                        adcigs[id] += s_P[iz+npml+nnz*(ix*dcdp+npml)+nnz*nnx*(iy*dcdp+npml)]
                                     *g_P[iz+npml+nnz*(ix*dcdp+npml)+nnz*nnx*(iy*dcdp+npml)]
                                     *cosf(ia*pi/180.0)*cosf(ia*pi/180.0)*cosf(ia*pi/180.0);
                     }
                 }
            }

        }
}
/*************func**************/
__global__ void adcigs_illum(int nnx, int nny, int nnz, int nx, int ny, int nz, int npml, int na, float *adcigs,int dcdp, float *illum)
{
    const int iz = blockIdx.x * blockDim.x + threadIdx.x;//0--nz's thread:iz
    const int ix = blockIdx.y * blockDim.y + threadIdx.y;//0--nx's thread:ix

       int id,iy,ia;

       for(iy=0;iy<(int)(ny/dcdp);iy++)
        {
          for(ia=0;ia<na;ia++)
           {

             if(ix<(int)(nx/dcdp)&&iz<nz)
               {

                        id=iz+ia*nz+ix*na*nz+iy*nz*na*((int)(nx/dcdp));
                        if(illum[iz+ix*dcdp*nz+iy*dcdp*nx*nz]!=0)
                             adcigs[id] /=illum[iz+ix*dcdp*nz+iy*dcdp*nx*nz];
               }
           }
       }
}
//a########################################################################
int main(int argc,char *argv[])
{
	int is, it, nx, ny, nz, nnx, nny, nnz, nt, wtype, na, da, dcdp, nxa, nya;
	int ns, nsx, dsx, fsx, dsy, fsy, zs, npml;
	float dx, dy, dz, dt, t, pfac, favg;

	float *v, *e, *d;
	float *vp, *epsilu, *deta;
	float *s_u0, *s_u1, *s_px0, *s_qx0, *s_px1, *s_qx1;
	float *s_v0, *s_v1, *s_py0, *s_qy0, *s_py1, *s_qy1;
       float *s_w0, *s_w1, *s_pz0, *s_qz0, *s_pz1, *s_qz1;
	float *s_P, *s_Q;
	float *g_u0, *g_u1, *g_px0, *g_qx0, *g_px1, *g_qx1;
	float *g_v0, *g_v1, *g_py0, *g_qy0, *g_py1, *g_qy1;
       float *g_w0, *g_w1, *g_pz0, *g_qz0, *g_pz1, *g_qz1;
	float *g_P, *g_Q;
       float *s_P_bndr, *s_Q_bndr;
       float *shot_Dev, *shot_Hos;
       float *migration, *illumination, *adcigs;
       float *Atemp;


       float *coffx1,*coffx2,*coffy1,*coffy2,*coffz1,*coffz2;
       float *acoffx1,*acoffx2,*acoffy1,*acoffy2,*acoffz1,*acoffz2;

       clock_t start, end, is_t0, is_t1;
/*************wavelet\boundary**************/
          wtype=1;npml=20;
/********** dat document ***********/
          char FN1[250]={"waxian_vel_201201201.dat"};
          char FN2[250]={"waxian_eps_201201201.dat"};
          char FN3[250]={"waxian_del_201201201.dat"};
	   char FN4[250]={"waxian_shot.dat"};
	   char FN5[250]={"waxian_snap.dat"};
	   char FN6[250]={"waxian_migration.dat"};
	   char FN7[250]={"waxian_migration_laplace.dat"};
	   char FN8[250]={"waxian_illumination.dat"};
	   char FN9[250]={"waxian_adcigs.dat"};

/********aaa************/
	 FILE *fpsnap, *fpshot, *fpmig,*fpmigla, *fpillum, *fpadcigs;
        fpshot=fopen(FN4,"wb");
        fpsnap=fopen(FN5,"wb");
        fpmig=fopen(FN6,"wb");
        fpmigla=fopen(FN7,"wb");
        fpillum=fopen(FN8,"wb");
        fpadcigs=fopen(FN9,"wb");


/********* parameters *************/

          nx=201;
          ny=201;
	   nz=201;         favg=60;     pfac=10.0;

 	   dx=5.0;
 	   dy=5.0;
          dz=5.0;

	   nt=1501;
          dt=0.0005;

          ns=625;          nsx=25;
         // fsx=nx/nsx/2;    dsx=nx/nsx;
         // fsy=ny/(ns/nsx)/2;    dsy=ny/(ns/nsx);
          fsx=4;//nx/ns/2;
          dsx=8;//nx/ns;
          fsy=4;//ny/ns/2;//200;//100;//ny/ns/2;
          dsy=8;//ny/ns;//0;//ny/ns;
          zs=1;

          na=65;
          da=1;
          dcdp=1;
/*************v***************/
          nnx=nx+2*npml;
          nny=ny+2*npml;
          nnz=nz+2*npml;
          nxa=(int)(nx/dcdp);
          nya=(int)(ny/dcdp);
/************a*************/
    	 Atemp=(float*)malloc(nz*nxa*nya*na*sizeof(float));

    	 v=(float*)malloc(nnz*nnx*nny*sizeof(float));
    	 e=(float*)malloc(nnz*nnx*nny*sizeof(float));
    	 d=(float*)malloc(nnz*nnx*nny*sizeof(float));
    	 shot_Hos=(float*)malloc(nt*nx*ny*sizeof(float));
        read_file(FN1,FN2,FN3,nx,ny,nz,nnx,nny,nnz,v,e,d,npml);
/****************************/
        pad_vv(nx,ny,nz,nnx,nny,nnz,npml,e);
        pad_vv(nx,ny,nz,nnx,nny,nnz,npml,d);
        pad_vv(nx,ny,nz,nnx,nny,nnz,npml,v);

        hipSetDevice(0);// initialize device, default device=0;
	 check_gpu_error("Failed to initialize device!");

	dim3 dimg, Xdimg, Adimg, dimb;
	dimg.x=(nnz+BlockSize1-1)/BlockSize1;
	dimg.y=(nnx+BlockSize2-1)/BlockSize2;
	Xdimg.x=(nnx+BlockSize1-1)/BlockSize1;
	Xdimg.y=(nny+BlockSize2-1)/BlockSize2;
	Adimg.x=(nz+BlockSize1-1)/BlockSize1;
	Adimg.y=(nxa+BlockSize2-1)/BlockSize2;
	dimb.x=BlockSize1;
	dimb.y=BlockSize2;
/****************************/
        hipMalloc(&vp, nnz*nnx*nny*sizeof(float));
        hipMalloc(&epsilu, nnz*nnx*nny*sizeof(float));
        hipMalloc(&deta, nnz*nnx*nny*sizeof(float));
	 hipMemcpy(vp, v, nnz*nnx*nny*sizeof(float), hipMemcpyHostToDevice);
	 hipMemcpy(epsilu, e, nnz*nnx*nny*sizeof(float), hipMemcpyHostToDevice);
	 hipMemcpy(deta, d, nnz*nnx*nny*sizeof(float), hipMemcpyHostToDevice);
/****************************/
        hipMalloc(&s_u0, nnz*nnx*nny*sizeof(float));    hipMalloc(&s_u1, nnz*nnx*nny*sizeof(float));
        hipMalloc(&s_v0, nnz*nnx*nny*sizeof(float));    hipMalloc(&s_v1, nnz*nnx*nny*sizeof(float));
        hipMalloc(&s_w0, nnz*nnx*nny*sizeof(float));    hipMalloc(&s_w1, nnz*nnx*nny*sizeof(float));

        hipMalloc(&s_P, nnz*nnx*nny*sizeof(float));     hipMalloc(&s_Q, nnz*nnx*nny*sizeof(float));

        hipMalloc(&s_px0, nnz*nnx*nny*sizeof(float));   hipMalloc(&s_px1, nnz*nnx*nny*sizeof(float));
        hipMalloc(&s_py0, nnz*nnx*nny*sizeof(float));   hipMalloc(&s_py1, nnz*nnx*nny*sizeof(float));
        hipMalloc(&s_pz0, nnz*nnx*nny*sizeof(float));   hipMalloc(&s_pz1, nnz*nnx*nny*sizeof(float));
        hipMalloc(&s_qx0, nnz*nnx*nny*sizeof(float));   hipMalloc(&s_qx1, nnz*nnx*nny*sizeof(float));
        hipMalloc(&s_qy0, nnz*nnx*nny*sizeof(float));   hipMalloc(&s_qy1, nnz*nnx*nny*sizeof(float));
        hipMalloc(&s_qz0, nnz*nnx*nny*sizeof(float));   hipMalloc(&s_qz1, nnz*nnx*nny*sizeof(float));

        hipMalloc(&g_u0, nnz*nnx*nny*sizeof(float));    hipMalloc(&g_u1, nnz*nnx*nny*sizeof(float));
        hipMalloc(&g_v0, nnz*nnx*nny*sizeof(float));    hipMalloc(&g_v1, nnz*nnx*nny*sizeof(float));
        hipMalloc(&g_w0, nnz*nnx*nny*sizeof(float));    hipMalloc(&g_w1, nnz*nnx*nny*sizeof(float));

        hipMalloc(&g_P, nnz*nnx*nny*sizeof(float));     hipMalloc(&g_Q, nnz*nnx*nny*sizeof(float));

        hipMalloc(&g_px0, nnz*nnx*nny*sizeof(float));   hipMalloc(&g_px1, nnz*nnx*nny*sizeof(float));
        hipMalloc(&g_py0, nnz*nnx*nny*sizeof(float));   hipMalloc(&g_py1, nnz*nnx*nny*sizeof(float));
        hipMalloc(&g_pz0, nnz*nnx*nny*sizeof(float));   hipMalloc(&g_pz1, nnz*nnx*nny*sizeof(float));
        hipMalloc(&g_qx0, nnz*nnx*nny*sizeof(float));   hipMalloc(&g_qx1, nnz*nnx*nny*sizeof(float));
        hipMalloc(&g_qy0, nnz*nnx*nny*sizeof(float));   hipMalloc(&g_qy1, nnz*nnx*nny*sizeof(float));
        hipMalloc(&g_qz0, nnz*nnx*nny*sizeof(float));   hipMalloc(&g_qz1, nnz*nnx*nny*sizeof(float));

        hipMalloc(&coffx1, nnx*sizeof(float));     hipMalloc(&coffx2, nnx*sizeof(float));
        hipMalloc(&coffy1, nnx*sizeof(float));     hipMalloc(&coffy2, nnx*sizeof(float));
        hipMalloc(&coffz1, nnz*sizeof(float));     hipMalloc(&coffz2, nnz*sizeof(float));
        hipMalloc(&acoffx1, nnx*sizeof(float));    hipMalloc(&acoffx2, nnx*sizeof(float));
        hipMalloc(&acoffy1, nnx*sizeof(float));    hipMalloc(&acoffy2, nnx*sizeof(float));
        hipMalloc(&acoffz1, nnz*sizeof(float));    hipMalloc(&acoffz2, nnz*sizeof(float));

        hipMalloc(&s_P_bndr, nt*(2*nx*ny+2*nz*ny+2*nx*nz)*sizeof(float));
        hipMalloc(&s_Q_bndr, nt*(2*nx*ny+2*nz*ny+2*nx*nz)*sizeof(float));

        hipMalloc(&migration, nz*nx*ny*sizeof(float));
        hipMalloc(&illumination, nz*nx*ny*sizeof(float));
        hipMalloc(&adcigs, nz*nxa*nya*na*sizeof(float));

        hipMalloc(&shot_Dev, nx*ny*nt*sizeof(float));
/******************************/
	 check_gpu_error("Failed to allocate memory for variables!");

        get_d0<<<1, 1>>>(dx,dy,dz,nnx,nny,nnz,npml,vp);
        initial_coffe<<<(nnx+511)/512, 512>>>(dt,nx,coffx1,coffx2,acoffx1,acoffx2,npml);
        initial_coffe<<<(nny+511)/512, 512>>>(dt,ny,coffy1,coffy2,acoffy1,acoffy2,npml);
        initial_coffe<<<(nnz+511)/512, 512>>>(dt,nz,coffz1,coffz2,acoffz1,acoffz2,npml);

        hipMemset(migration, 0, nz*nx*ny*sizeof(float));
        hipMemset(illumination, 0, nz*nx*ny*sizeof(float));
        hipMemset(adcigs, 0, nz*na*nxa*nya*sizeof(float));

        printf("--------------------------------------------------------\n");
        printf("---   \n");
        start = clock();
/**********IS Loop start*******/
   for(is=0;is<ns;is++)
    {
       //  printf("---   IS=%3d  \n",is);
     is_t0 = clock();

     hipMemset(s_u0, 0, nnz*nnx*nny*sizeof(float));     hipMemset(s_u1, 0, nnz*nnx*nny*sizeof(float));
     hipMemset(s_v0, 0, nnz*nnx*nny*sizeof(float));     hipMemset(s_v1, 0, nnz*nnx*nny*sizeof(float));
     hipMemset(s_w0, 0, nnz*nnx*nny*sizeof(float));     hipMemset(s_w1, 0, nnz*nnx*nny*sizeof(float));

     hipMemset(s_P, 0, nnz*nnx*nny*sizeof(float));      hipMemset(s_Q, 0, nnz*nnx*nny*sizeof(float));

     hipMemset(s_px0, 0, nnz*nnx*nny*sizeof(float));    hipMemset(s_px1, 0, nnz*nnx*nny*sizeof(float));
     hipMemset(s_py0, 0, nnz*nnx*nny*sizeof(float));    hipMemset(s_py1, 0, nnz*nnx*nny*sizeof(float));
     hipMemset(s_pz0, 0, nnz*nnx*nny*sizeof(float));    hipMemset(s_pz1, 0, nnz*nnx*nny*sizeof(float));
     hipMemset(s_qx0, 0, nnz*nnx*nny*sizeof(float));    hipMemset(s_qx1, 0, nnz*nnx*nny*sizeof(float));
     hipMemset(s_qy0, 0, nnz*nnx*nny*sizeof(float));    hipMemset(s_qy1, 0, nnz*nnx*nny*sizeof(float));
     hipMemset(s_qz0, 0, nnz*nnx*nny*sizeof(float));    hipMemset(s_qz1, 0, nnz*nnx*nny*sizeof(float));

     hipMemset(g_u0, 0, nnz*nnx*nny*sizeof(float));     hipMemset(g_u1, 0, nnz*nnx*nny*sizeof(float));
     hipMemset(g_v0, 0, nnz*nnx*nny*sizeof(float));     hipMemset(g_v1, 0, nnz*nnx*nny*sizeof(float));
     hipMemset(g_w0, 0, nnz*nnx*nny*sizeof(float));     hipMemset(g_w1, 0, nnz*nnx*nny*sizeof(float));

     hipMemset(g_P, 0, nnz*nnx*nny*sizeof(float));      hipMemset(g_Q, 0, nnz*nnx*nny*sizeof(float));

     hipMemset(g_px0, 0, nnz*nnx*nny*sizeof(float));    hipMemset(g_px1, 0, nnz*nnx*nny*sizeof(float));
     hipMemset(g_py0, 0, nnz*nnx*nny*sizeof(float));    hipMemset(g_py1, 0, nnz*nnx*nny*sizeof(float));
     hipMemset(g_pz0, 0, nnz*nnx*nny*sizeof(float));    hipMemset(g_pz1, 0, nnz*nnx*nny*sizeof(float));
     hipMemset(g_qx0, 0, nnz*nnx*nny*sizeof(float));    hipMemset(g_qx1, 0, nnz*nnx*nny*sizeof(float));
     hipMemset(g_qy0, 0, nnz*nnx*nny*sizeof(float));    hipMemset(g_qy1, 0, nnz*nnx*nny*sizeof(float));
     hipMemset(g_qz0, 0, nnz*nnx*nny*sizeof(float));    hipMemset(g_qz1, 0, nnz*nnx*nny*sizeof(float));

     hipMemset(s_P_bndr, 0, nt*(2*nx*ny+2*nz*ny+2*nx*nz)*sizeof(float));
     hipMemset(s_Q_bndr, 0, nt*(2*nx*ny+2*nz*ny+2*nx*nz)*sizeof(float));

     hipMemset(shot_Dev, 0, nt*nx*ny*sizeof(float));

     for(it=0,t=dt;it<nt;it++,t+=dt)
     {
      if(it%100==0)printf("---   IS===%d   it===%d\n",is,it);
        add_source<<<1,1>>>(pfac,fsx,fsy,zs,nx,ny,nz,nnx,nny,nnz,dt,t,favg,wtype,npml,is,dsx,dsy,s_P,s_Q,nsx);
        update_vel<<<dimg,dimb>>>(nx,ny,nz,nnx,nny,nnz,npml,dt,dx,dy,dz,
                                 s_u0,s_v0,s_w0,s_u1,s_v1,s_w1,s_P,s_Q,coffx1,coffx2,coffy1,coffy2,coffz1,coffz2);
        update_stress<<<dimg,dimb>>>(nx,ny,nz,nnx,nny,nnz,dt,dx,dy,dz,s_u1,s_v1,s_w1,s_P,s_Q,vp,npml,
                                     s_px1,s_px0,s_py1,s_py0,s_pz1,s_pz0,s_qx1,s_qx0,s_qy1,s_qy0,s_qz1,s_qz0,
                                     acoffx1,acoffx2,acoffy1,acoffy2,acoffz1,acoffz2,deta,epsilu,
                                     fsx, dsx, fsy, dsy,zs, is, nsx, true);
        s_u0=s_u1; s_v0=s_v1; s_w0=s_w1; s_px0=s_px1; s_py0=s_py1; s_pz0=s_pz1; s_qx0=s_qx1; s_qy0=s_qy1; s_qz0=s_qz1;

        wavefield_bndr<<<(2*nx*ny+2*nz*ny+2*nx*nz+511)/512, 512>>>(nnx,nny,nnz,nx,ny,nz,npml,it,nt,s_P,s_Q,s_P_bndr,s_Q_bndr,true);
        shot_record<<<(nx*ny+511)/512, 512>>>(nnx,nny, nnz, nx, ny, nz, npml, it, nt, s_P, s_Q, shot_Dev, true);

        cal_illumination<<<dimg,dimb>>>(nnx,nny,nnz,nx,ny,nz,npml,illumination,s_P,s_Q);

    /*       if((is==0)&&(it!=0&&it%100==0))
            {
	       hipMemcpy(e, s_P, nnz*nnx*nny*sizeof(float), hipMemcpyDeviceToHost);
            //  fseek(fpsnap,(int)(it/100-1)*(nx*ny*nz)*4L,0);
              window3d(v, e, nz, nx, ny, nnz, nnx, npml);
              fwrite(v,4L,nx*nz*ny,fpsnap);
            }   */
     }//it loop end

      mute_directwave<<<Xdimg,dimb>>>(nx,ny,nt,dt,favg,dx,dy,dz,fsx,fsy,dsx,dsy,zs,is,vp,epsilu,shot_Dev,70,nsx);

   //   if(is==0){
          hipMemcpy(shot_Hos, shot_Dev, nt*nx*ny*sizeof(float), hipMemcpyDeviceToHost);
         // fseek(fpshot,is*nt*nx*ny*sizeof(float),0);
          fwrite(shot_Hos,sizeof(float),nt*nx*ny,fpshot);
    //   }

    for(it=nt-1;it>=0;it--)
     {
      if(it%100==0)printf("---   IS===%d   it===%d\n",is,it);
        wavefield_bndr<<<(2*nx*ny+2*nz*ny+2*nx*nz+511)/512, 512>>>(nnx,nny,nnz,nx,ny,nz,npml,it,nt,s_P,s_Q,s_P_bndr,s_Q_bndr,false);
        update_vel<<<dimg,dimb>>>(nx,ny,nz,nnx,nny,nnz,npml,dt,dx,dy,dz,
                                 s_u0,s_v0,s_w0,s_u1,s_v1,s_w1,s_P,s_Q,coffx1,coffx2,coffy1,coffy2,coffz1,coffz2);
        update_stress<<<dimg,dimb>>>(nx,ny,nz,nnx,nny,nnz,dt,dx,dy,dz,s_u1,s_v1,s_w1,s_P,s_Q,vp,npml,
                                     s_px1,s_px0,s_py1,s_py0,s_pz1,s_pz0,s_qx1,s_qx0,s_qy1,s_qy0,s_qz1,s_qz0,
                                     acoffx1,acoffx2,acoffy1,acoffy2,acoffz1,acoffz2,deta,epsilu,
                                     fsx, dsx, fsy, dsy,zs, is, nsx, true);
        s_u0=s_u1; s_v0=s_v1; s_w0=s_w1; s_px0=s_px1; s_py0=s_py1; s_pz0=s_pz1; s_qx0=s_qx1; s_qy0=s_qy1; s_qz0=s_qz1;

      /*     if((is==0)&&(it!=0&&it%100==0))
            {
	       hipMemcpy(e, s_P, nnz*nnx*nny*sizeof(float), hipMemcpyDeviceToHost);
             // fseek(fpsnap,(int)(it/100-1)*(nx*ny*nz)*4L,0);
              window3d(v, e, nz, nx, ny, nnz, nnx, npml);
              fwrite(v,4L,nx*nz*ny,fpsnap);
            }    */

        shot_record<<<(nx*ny+511)/512, 512>>>(nnx,nny, nnz, nx, ny, nz, npml, it, nt, g_P, g_Q, shot_Dev, false);
        update_vel<<<dimg,dimb>>>(nx,ny,nz,nnx,nny,nnz,npml,dt,dx,dy,dz,
                                 g_u0,g_v0,g_w0,g_u1,g_v1,g_w1,g_P,g_Q,coffx1,coffx2,coffy1,coffy2,coffz1,coffz2);
        update_stress<<<dimg,dimb>>>(nx,ny,nz,nnx,nny,nnz,dt,dx,dy,dz,g_u1,g_v1,g_w1,g_P,g_Q,vp,npml,
                                     g_px1,g_px0,g_py1,g_py0,g_pz1,g_pz0,g_qx1,g_qx0,g_qy1,g_qy0,g_qz1,g_qz0,
                                     acoffx1,acoffx2,acoffy1,acoffy2,acoffz1,acoffz2,deta,epsilu,
                                     fsx, dsx, fsy, dsy,zs, is, nsx, true);
        g_u0=g_u1; g_v0=g_v1; g_w0=g_w1; g_px0=g_px1; g_py0=g_py1; g_pz0=g_pz1; g_qx0=g_qx1; g_qy0=g_qy1; g_qz0=g_qz1;

    /*       if((is==0)&&(it!=0&&it%100==0))
            {
	       hipMemcpy(e, g_P, nnz*nnx*nny*sizeof(float), hipMemcpyDeviceToHost);
            //  fseek(fpsnap,(int)(it/100-1)*(nx*ny*nz)*4L,0);
              window3d(v, e, nz, nx, ny, nnz, nnx, npml);
              fwrite(v,4L,nx*nz*ny,fpsnap);
            }    */
        cal_illumination<<<dimg,dimb>>>(nnx,nny,nnz,nx,ny,nz,npml,illumination,g_P,g_Q);
        cal_migration<<<dimg,dimb>>>(nnx,nny,nnz,nx,ny,nz,npml,migration,s_P,g_P);

        Poynting_Adcigs<<<Adimg,dimb>>>(nnx,nny,nnz,nx,ny,nz,npml,na,da,adcigs,dcdp,
                                                s_P,s_Q,s_u0,s_v0,s_w0,g_P,g_Q,g_u0,g_v0,g_w0);

     }//it loop end
    is_t1 = clock();
    printf("IS=%3d: %f (min)\n", is, ((float)(is_t1-is_t0))/60.0/CLOCKS_PER_SEC);
    }//is loop end

   migration_illum<<<dimg,dimb>>>(nnx,nny,nnz,nx,ny,nz,npml,illumination,migration);
   adcigs_illum<<<Adimg,dimb>>>(nnx,nny,nnz,nx,ny,nz,npml,na,adcigs,dcdp,illumination);

   hipMemcpy(v, illumination, nz*nx*ny*sizeof(float), hipMemcpyDeviceToHost);
   fwrite(v,4L,nx*nz*ny,fpillum);

   hipMemcpy(v, migration, nz*nx*ny*sizeof(float), hipMemcpyDeviceToHost);
   fwrite(v,4L,nx*nz*ny,fpmig);
   laplace_3d_filter(1, nz, nx, ny, v, e);
   fwrite(e,4L,nx*nz*ny,fpmigla);

   hipMemcpy(Atemp, adcigs, nz*nxa*nya*na*sizeof(float), hipMemcpyDeviceToHost);
   fwrite(Atemp,sizeof(float),nz*nxa*nya*na,fpadcigs);


    end = clock();
/*********IS Loop end*********/
   printf("---   The forward is over    \n");
   printf("---   Complete!!!!!!!!! \n");
   printf("total %d shots: %f (min)\n", ns, ((float)(end-start))/60.0/CLOCKS_PER_SEC);



/***********close************/
          fclose(fpsnap);   fclose(fpshot);   fclose(fpmig);fclose(fpmigla); fclose(fpillum); fclose(fpadcigs);
/***********free*************/
       hipFree(coffx1);       hipFree(coffx2);
       hipFree(coffz1);       hipFree(coffz2);
       hipFree(acoffx1);      hipFree(acoffx2);
       hipFree(acoffz1);      hipFree(acoffz2);

       hipFree(s_u0);           hipFree(s_u1);
       hipFree(s_v0);           hipFree(s_v1);
       hipFree(s_w0);           hipFree(s_w1);

       hipFree(s_P);            hipFree(s_Q);

       hipFree(s_px0);          hipFree(s_px1);
       hipFree(s_py0);          hipFree(s_py1);
       hipFree(s_pz0);          hipFree(s_pz1);
       hipFree(s_qx0);          hipFree(s_qx1);
       hipFree(s_qy0);          hipFree(s_qy1);
       hipFree(s_qz0);          hipFree(s_qz1);

       hipFree(g_u0);           hipFree(g_u1);
       hipFree(g_v0);           hipFree(g_v1);
       hipFree(g_w0);           hipFree(g_w1);

       hipFree(g_P);            hipFree(g_Q);

       hipFree(g_px0);          hipFree(g_px1);
       hipFree(g_py0);          hipFree(g_py1);
       hipFree(g_pz0);          hipFree(g_pz1);
       hipFree(g_qx0);          hipFree(g_qx1);
       hipFree(g_qy0);          hipFree(g_qy1);
       hipFree(g_qz0);          hipFree(g_qz1);

       hipFree(s_P_bndr);
       hipFree(s_Q_bndr);

       hipFree(shot_Dev);

       hipFree(migration);
       hipFree(illumination);
       hipFree(adcigs);
/***************host free*****************/
	free(v);	free(e);	free(d);
       free(shot_Hos);  free(Atemp);
}

